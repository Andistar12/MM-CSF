#include "hip/hip_runtime.h"
#include <iostream>
#include "mttkrp_gpu.h"
#include <vector>

inline hipError_t checkCuda(hipError_t result, int s){

  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error in line : %s - %d\n", hipGetErrorString(result), s);
    assert(result == hipSuccess);
  }
  return result;
}

void cuda_timer_start(hipEvent_t start){
	checkCuda(hipEventRecord(start), __LINE__);
}
void cuda_timer_stop(hipEvent_t start, hipEvent_t stop, float &mili){
	checkCuda(hipEventRecord(stop), __LINE__);
    hipEventSynchronize(stop);
    checkCuda(hipEventElapsedTime(&mili, start, stop), __LINE__);
    hipDeviceSynchronize();
}

// CUDA kernel call to do COO MTTKRP 
__global__ void mttkrp_COO_kernel(DTYPE *vals, ITYPE *dInds0, ITYPE *dInds1, ITYPE *dInds2,  ITYPE nnz,
	DTYPE *dU0, DTYPE *dU1, DTYPE *dU2, ITYPE	mode, ITYPE R){

	unsigned int tId = threadIdx.x;
	unsigned int laneId = tId & 31;
	unsigned int gId = (blockIdx.x * blockDim.x + tId);
	unsigned int x = gId >> 5;
	
	if(x < nnz){
        DTYPE tmp_val = 0;
        ITYPE idx0 = dInds0[x];
        ITYPE idx1 = dInds1[x];
        ITYPE idx2 = dInds2[x];

        for(ITYPE r=laneId; r<R; r+=32) {           
            tmp_val = vals[x] * dU1[idx1 * R + r] * dU2[idx2 * R + r];
            atomicAdd(&dU0[idx0 * R + r], tmp_val);
        }    
	}
}

// CUDA kernel call to do COO MTTKRP using loop
__global__ void mttkrp_COO_kernel_loop(DTYPE * const vals, ITYPE * const dInds0, ITYPE * const dInds1, ITYPE * const dInds2,  const ITYPE nnz,
	DTYPE *dU0, DTYPE * const dU1, DTYPE * const dU2, ITYPE	mode, ITYPE R){

	unsigned int tId = threadIdx.x;
	unsigned int laneId = tId & 31;
	unsigned int gId = (blockIdx.x * blockDim.x + tId);

	//like PARTI
	size_t num_loops_nnz = 1 * 32;
    size_t const nnz_per_loop = gridDim.x * blockDim.x;
    if(nnz > nnz_per_loop) {
        num_loops_nnz = ((nnz + nnz_per_loop - 1) / nnz_per_loop) << 5;
    }

	unsigned int x;

	for(size_t nl=0; nl<num_loops_nnz; ++nl) {
		
		x = (gId + nl * nnz_per_loop) >> 5;
		
		if(x < nnz){
	    
	        DTYPE tmp_val = 0;
	        ITYPE idx0 = dInds0[x];
	        ITYPE idx1 = dInds1[x];
	        ITYPE idx2 = dInds2[x];

	        for(ITYPE r=laneId; r<R; r+=32) {           
	            tmp_val = vals[x] * dU1[idx1 * R + r] * dU2[idx2 * R + r];
	            atomicAdd(&dU0[idx0 * R + r], tmp_val);
	        }  
		}
		__syncthreads();
	}
}
// CUDA kernel call to do COO MTTKRP 4D 
__global__ void mttkrp_COO_kernel_4D(DTYPE *vals, ITYPE *dInds0, ITYPE *dInds1, ITYPE *dInds2, ITYPE *dInds3,
    ITYPE nnz, DTYPE *dU0, DTYPE *dU1, DTYPE *dU2,  DTYPE *dU3, ITYPE mode, ITYPE R){

	unsigned int tId = threadIdx.x;
	unsigned int laneId = tId & 31;
	unsigned int gId = (blockIdx.x * blockDim.x + tId);
	unsigned int x = gId >> 5;
	
	if(x < nnz){
        DTYPE tmp_val = 0;
        ITYPE idx0 = dInds0[x];
        ITYPE idx1 = dInds1[x];
        ITYPE idx2 = dInds2[x];
        ITYPE idx3 = dInds3[x];

        for(ITYPE r=laneId; r<R; r+=32) {           
            tmp_val = vals[x] * dU1[idx1 * R + r] * dU2[idx2 * R + r]  * dU3[idx3 * R + r];
            atomicAdd(&dU0[idx0 * R + r], tmp_val);
        }    
	}
}

// CUDA kernel call to do COO MTTKRP 4D using loop
__global__ void mttkrp_COO_kernel_4D_loop(DTYPE *const vals, ITYPE * const dInds0, ITYPE * const dInds1, ITYPE *const dInds2, ITYPE * const dInds3,
    ITYPE nnz, DTYPE *dU0, DTYPE * const dU1, DTYPE * const dU2,  DTYPE * const dU3, ITYPE mode, ITYPE R){

	unsigned int tId = threadIdx.x;
	unsigned int laneId = tId & 31;
	unsigned int gId = (blockIdx.x * blockDim.x + tId);
	
	//like PARTI
	size_t num_loops_nnz = 1 * 32;
    size_t const nnz_per_loop = gridDim.x * blockDim.x;
    if(nnz > nnz_per_loop) {
        num_loops_nnz = ((nnz + nnz_per_loop - 1) / nnz_per_loop) << 5;
    }
	unsigned int x;

	for(size_t nl=0; nl<num_loops_nnz; ++nl) 
	{
		x = (gId + nl * nnz_per_loop) >> 5;

		if(x < nnz){
	        DTYPE tmp_val = 0;
	        ITYPE idx0 = dInds0[x];
	        ITYPE idx1 = dInds1[x];
	        ITYPE idx2 = dInds2[x];
	        ITYPE idx3 = dInds3[x];

	        for(ITYPE r=laneId; r<R; r+=32) {           
	            tmp_val = vals[x] * dU1[idx1 * R + r] * dU2[idx2 * R + r]  * dU3[idx3 * R + r];
	            atomicAdd(&dU0[idx0 * R + r], tmp_val);
	        }
	    }  
	    __syncthreads();  
	}
}
//no atomics because all 1 in HYB - COO 
__global__ void mttkrp_HYB_COO_kernel(DTYPE *vals, ITYPE *dInds0, ITYPE *dInds1, ITYPE *dInds2,  ITYPE nnz,
	DTYPE *dU0, DTYPE *dU1, DTYPE *dU2, ITYPE	mode, ITYPE R){

	unsigned int tId = threadIdx.x;
	unsigned int laneId = tId & 31;
	unsigned int gId = (blockIdx.x * blockDim.x + tId);
	unsigned int x = gId >> 5;
	
	if(x < nnz){
        DTYPE tmp_val = 0;
        ITYPE idx0 = dInds0[x];
        ITYPE idx1 = dInds1[x];
        ITYPE idx2 = dInds2[x];

        for(ITYPE r=laneId; r<R; r+=32) {           
            tmp_val = vals[x] * dU1[idx1 * R + r] * dU2[idx2 * R + r];
            dU0[idx0 * R + r] += tmp_val;
        }    
	}
}

// CUDA kernel call to do COO MTTKRP using loop
__global__ void mttkrp_HYB_COO_kernel_loop(DTYPE * const vals, ITYPE * const dInds0, ITYPE * const dInds1, ITYPE * const dInds2,  const ITYPE nnz,
	DTYPE *dU0, DTYPE * const dU1, DTYPE * const dU2, ITYPE	mode, ITYPE R){

	unsigned int tId = threadIdx.x;
	unsigned int laneId = tId & 31;
	unsigned int gId = (blockIdx.x * blockDim.x + tId);

	//like PARTI
	size_t num_loops_nnz = 1 * 32;
    size_t const nnz_per_loop = gridDim.x * blockDim.x;
    if(nnz > nnz_per_loop) {
        num_loops_nnz = ((nnz + nnz_per_loop - 1) / nnz_per_loop) << 5;
    }

	unsigned int x;

	for(size_t nl=0; nl<num_loops_nnz; ++nl) {
		
		x = (gId + nl * nnz_per_loop) >> 5;
		
		if(x < nnz){
	    
	        DTYPE tmp_val = 0;
	        ITYPE idx0 = dInds0[x];
	        ITYPE idx1 = dInds1[x];
	        ITYPE idx2 = dInds2[x];

	        for(ITYPE r=laneId; r<R; r+=32) {           
	            tmp_val = vals[x] * dU1[idx1 * R + r] * dU2[idx2 * R + r];
	            dU0[idx0 * R + r] += tmp_val;
	        }  
		}
		__syncthreads();
	}
}

//no atomics because all 1 in HYB - COO 
__global__ void mttkrp_HYB_COO_kernel_4D(DTYPE *vals, ITYPE *dInds0, ITYPE *dInds1, ITYPE *dInds2, ITYPE *dInds3,
  ITYPE nnz,  DTYPE *dU0, DTYPE *dU1, DTYPE *dU2,  DTYPE *dU3, ITYPE mode, ITYPE R){

	unsigned int tId = threadIdx.x;
	unsigned int laneId = tId & 31;
	unsigned int gId = (blockIdx.x * blockDim.x + tId);
	unsigned int x = gId >> 5;
	
	if(x < nnz){
        DTYPE tmp_val = 0;
        ITYPE idx0 = dInds0[x];
        ITYPE idx1 = dInds1[x];
        ITYPE idx2 = dInds2[x];
        ITYPE idx3 = dInds3[x];

        for(ITYPE r=laneId; r<R; r+=32) {           
            tmp_val = vals[x] * dU1[idx1 * R + r] * dU2[idx2 * R + r] * dU3[idx3 * R + r];
            dU0[idx0 * R + r] += tmp_val;
        }    
	}
}

// CUDA kernel call to do COO MTTKRP 4D using loop
__global__ void mttkrp_HYB_COO_kernel_4D_loop(DTYPE *const vals, ITYPE * const dInds0, ITYPE * const dInds1, ITYPE *const dInds2, ITYPE * const dInds3,
    ITYPE nnz, DTYPE *dU0, DTYPE * const dU1, DTYPE * const dU2,  DTYPE * const dU3, ITYPE mode, ITYPE R){

	unsigned int tId = threadIdx.x;
	unsigned int laneId = tId & 31;
	unsigned int gId = (blockIdx.x * blockDim.x + tId);
	
	//like PARTI
	size_t num_loops_nnz = 1 * 32;
    size_t const nnz_per_loop = gridDim.x * blockDim.x;
    if(nnz > nnz_per_loop) {
        num_loops_nnz = ((nnz + nnz_per_loop - 1) / nnz_per_loop) << 5;
    }
	unsigned int x;

	for(size_t nl=0; nl<num_loops_nnz; ++nl) 
	{
		x = (gId + nl * nnz_per_loop) >> 5;

		if(x < nnz){
	        DTYPE tmp_val = 0;
	        ITYPE idx0 = dInds0[x];
	        ITYPE idx1 = dInds1[x];
	        ITYPE idx2 = dInds2[x];
	        ITYPE idx3 = dInds3[x];

	        for(ITYPE r=laneId; r<R; r+=32) {           
	            tmp_val = vals[x] * dU1[idx1 * R + r] * dU2[idx2 * R + r]  * dU3[idx3 * R + r];
	            dU0[idx0 * R + r] += tmp_val;
	        }
	    }  
	    __syncthreads();  
	}
}

__global__ void mttkrp_CSL_kernel(DTYPE * vals, ITYPE *dfbrIdx0, ITYPE *dSlcMapperBin, ITYPE *dInds2, ITYPE *fbrPtr0,
	ITYPE *dInds1, unsigned int nSlices, DTYPE *dU0, DTYPE * dU1, DTYPE *dU2, 
	ITYPE mode, ITYPE R, ITYPE warpPerSlice, int logOfWPC, int TbPerSlc, int LogOfTPS){

	unsigned int tId = threadIdx.x;
	unsigned int laneId = tId & 31;
	unsigned int gId = (blockIdx.x * blockDim.x + tId);
	unsigned int workId = (tId & ((1 << (5 + logOfWPC)) - 1)) >> 5;  
	unsigned int slc = gId >> (5 + logOfWPC); // 5: minimum 1 WARP (2^5) 
	DTYPE tmp_val;
		              	              
	if(slc < nSlices){ 	    

		unsigned int mappedSlc = slc;//dSlcMapperBin[slc];
		unsigned int idx0 = dfbrIdx0[mappedSlc]; 
    	int fb_st = fbrPtr0[mappedSlc];
		int fb_end = fbrPtr0[mappedSlc+1];
		tmp_val = 0;
		
		for (int fbr = fb_st + workId; fbr < fb_end; fbr+=warpPerSlice){
			
		    unsigned int idx1 = dInds1[fbr];
	        unsigned int idx2 = dInds2[fbr];                
            for(unsigned int r=laneId; r<R; r+=32) {
                tmp_val += vals[fbr] * dU2[idx2 * R + r] * dU1[idx1 * R + r]; 
            }   
		}
		for(unsigned int r=laneId; r<R; r+=32) {  
			atomicAdd(&dU0[idx0 * R + r], tmp_val);    
		}
	}
}

__global__ void mttkrp_CSL_kernel_bin(DTYPE * vals, ITYPE *dfbrIdx0, ITYPE *dSlcMapperBin, ITYPE *dInds2, ITYPE *fbrPtr0,
	ITYPE *dInds1, unsigned int nSlices, DTYPE *dU0, DTYPE * dU1, DTYPE *dU2, 
	ITYPE mode, ITYPE R, ITYPE warpPerSlice, int logOfWPC){

	unsigned int tId = threadIdx.x;
	unsigned int laneId = tId & 31;
	unsigned int gId = (blockIdx.x * blockDim.x + tId);
	unsigned int workId = (tId & ((1 << (5 + logOfWPC)) - 1)) >> 5;  
	unsigned int slc = gId >> (5 + logOfWPC); // 5: minimum 1 WARP (2^5) 
	DTYPE tmp_val;
		              	              
	if(slc < nSlices){ 	    

		unsigned int mappedSlc = dSlcMapperBin[slc];
		unsigned int idx0 = dfbrIdx0[mappedSlc]; 
    	int fb_st = fbrPtr0[mappedSlc];
		int fb_end = fbrPtr0[mappedSlc+1];
		tmp_val = 0;
		
		for (int fbr = fb_st + workId; fbr < fb_end; fbr+=warpPerSlice){
			
		    unsigned int idx1 = dInds1[fbr];
	        unsigned int idx2 = dInds2[fbr];                
            for(unsigned int r=laneId; r<R; r+=32) {
                tmp_val += vals[fbr] * dU2[idx2 * R + r] * dU1[idx1 * R + r]; 
            }   
		}
		for(unsigned int r=laneId; r<R; r+=32) {  
			atomicAdd(&dU0[idx0 * R + r], tmp_val);    
		}
	}
}

// CSL kernel with loop like ParTI
__global__ void mttkrp_CSL_kernel_bin_loop(DTYPE * vals, ITYPE *dfbrIdx0, ITYPE *dSlcMapperBin, ITYPE *dInds2, ITYPE *fbrPtr0,
	ITYPE *dInds1, unsigned int nSlices, DTYPE *dU0, DTYPE * dU1, DTYPE *dU2, 
	ITYPE mode, ITYPE R, ITYPE warpPerSlice, int logOfWPC, int TbPerSlc, int LogOfTPS){

	unsigned int tId = threadIdx.x;
	unsigned int laneId = tId & 31;
	unsigned int gId = (blockIdx.x * blockDim.x + tId);
	unsigned int workId = (tId & ((1 << (5 + logOfWPC)) - 1)) >> 5;  
	unsigned int slc = gId >> (5 + logOfWPC); // 5: minimum 1 WARP (2^5) 
	DTYPE tmp_val;

	//like PARTI
	size_t num_loops_nnz = 1 * 32;
    size_t const nnz_per_loop = gridDim.x * blockDim.x;
    if(nSlices > nnz_per_loop) {
        num_loops_nnz = ((nSlices + nnz_per_loop - 1) / nnz_per_loop) << 5;
    }

	for(size_t nl=0; nl<num_loops_nnz; ++nl) {
		
		slc = (gId + nl * nnz_per_loop) >> 5;
		              	              
		if(slc < nSlices){ 	    

			unsigned int mappedSlc = dSlcMapperBin[slc];
			unsigned int idx0 = dfbrIdx0[mappedSlc]; 
	    	int fb_st = fbrPtr0[mappedSlc];
			int fb_end = fbrPtr0[mappedSlc+1];
			tmp_val = 0;
			
			for (int fbr = fb_st + workId; fbr < fb_end; fbr+=warpPerSlice){
				
			    unsigned int idx1 = dInds1[fbr];
		        unsigned int idx2 = dInds2[fbr];                
	            for(unsigned int r=laneId; r<R; r+=32) {
	                tmp_val += vals[fbr] * dU2[idx2 * R + r] * dU1[idx1 * R + r]; 
	            }   
			}
			for(unsigned int r=laneId; r<R; r+=32) {  
				atomicAdd(&dU0[idx0 * R + r], tmp_val);    
			}
		}
		__syncthreads();  
	}
}

// CUDA kernel call to do HCSR MTTKRP 
__global__ void mttkrp_CSL_kernel_hvyBin(DTYPE * vals, ITYPE *dfbrIdx0, ITYPE *dSlcMapperBin, ITYPE *dInds2, ITYPE *fbrPtr0,
	ITYPE *dInds1, unsigned int nSlices, DTYPE *dU0, DTYPE * dU1, DTYPE *dU2, 
	ITYPE	mode, ITYPE R, ITYPE warpPerSlice, int logOfWPC, int TbPerSlc, int logOfTPS){
	
	unsigned int laneId = threadIdx.x & 31;
	unsigned int workId = threadIdx.x >> 5;
	unsigned int slc = blockIdx.x >> logOfTPS;
	unsigned int localBId = blockIdx.x & (TbPerSlc -1);
	
	DTYPE tmp = 0, tmp_val;
		              	              
	if(slc < nSlices){

		unsigned int mappedSlc = dSlcMapperBin[slc];
		unsigned int idx0 = dfbrIdx0[mappedSlc] ;//slc;
		unsigned int nFbr = fbrPtr0[mappedSlc+1] - fbrPtr0[mappedSlc];		
		unsigned int fbrPerTb = (nFbr + TbPerSlc - 1 ) >> logOfTPS; 
		unsigned int fb_st = fbrPtr0[mappedSlc] + localBId * fbrPerTb ;
		unsigned int fb_end = fbrPtr0[mappedSlc] + (localBId + 1) * fbrPerTb ;

		tmp_val = 0;
		for (int fbr = fb_st + workId; fbr < fb_end && fbr < fbrPtr0[mappedSlc+1]; fbr+=warpPerSlice){
			
		    unsigned int idx1 = dInds1[fbr];
	        unsigned int idx2 = dInds2[fbr];                
            for(unsigned int r=laneId; r<R; r+=32) {
                tmp_val += vals[fbr] * dU2[idx2 * R + r] * dU1[idx1 * R + r]; 
            }   
		}
		for(unsigned int r=laneId; r<R; r+=32) {  
			atomicAdd(&dU0[idx0 * R + r], tmp_val);    
		} 
	}
}

// HCSR MTTKRP : 16 WARP = 1 TB per slice
__global__ void mttkrp_HCSR_kernel_16WARP(DTYPE * vals, ITYPE *dfbrIdx0, ITYPE *dSlcMapperBin, ITYPE *dInds2, ITYPE *fbrPtr0,
	ITYPE *fbrPtr1, ITYPE *fbrIdx1, unsigned int nSlices, DTYPE *dU0, DTYPE * dU1, DTYPE *dU2, 
	ITYPE	mode, ITYPE R, ITYPE warpPerSlice, int logOfWPC){

	unsigned int tId = threadIdx.x;
	unsigned int laneId = tId & 31;
	unsigned int gId = (blockIdx.x * blockDim.x + tId);
	unsigned int workId = tId >> 5; //(tId & ((1 << (5 + logOfWPC)) - 1)) >> 5;  
	unsigned int slc = blockIdx.x ;//gId >> (5 + logOfWPC); // 5: minimum 1 WARP (2^5) 
	DTYPE tmp = 0; 
	DTYPE tmp_val;
		              	              
	if(slc < nSlices){ 	    

		unsigned int mappedSlc = dSlcMapperBin[slc];
		unsigned int idx0 = dfbrIdx0[mappedSlc] ;//slc;
    	int fb_st = fbrPtr0[mappedSlc];
		int fb_end = fbrPtr0[mappedSlc+1];

		for (int fbr = fb_st + workId; fbr < fb_end; fbr+=warpPerSlice){
			tmp_val = 0;
	        for(unsigned int x = fbrPtr1[fbr]; x < fbrPtr1[fbr+1]; ++x) {

		        unsigned int idx2 = dInds2[x];                
	            for(unsigned int r=laneId; r<R; r+=32) {
	                tmp_val += vals[x] * dU2[idx2 * R + r]; 
	            }
	        }
	        // unsigned int idx1 = dInds1[fbrPtr1[fbr]]; 
	        unsigned int idx1 = fbrIdx1[fbr];   
	        for(unsigned int r=laneId; r<R; r+=32) {  
	        	tmp += tmp_val * dU1[idx1 * R + r] ;     
	        }    
		}

		for(unsigned int r=laneId; r<R; r+=32) {  
            atomicAdd(&dU0[idx0 * R + r], tmp);
        } 
	}
}
// CUDA kernel call to do HCSR MTTKRP for the first bin 1 WARP per slice
__global__ void mttkrp_HCSR_kernel_COO(DTYPE * vals, ITYPE *dfbrIdx0, ITYPE *dSlcMapperBin, ITYPE *dInds2, ITYPE *fbrPtr0,
	ITYPE *fbrPtr1, ITYPE *fbrIdx1, unsigned int nSlices, DTYPE *dU0, DTYPE * dU1, DTYPE *dU2, 
	ITYPE	mode, ITYPE R, ITYPE warpPerSlice, int logOfWPC, int TbPerSlc, int LogOfTPS){

	unsigned int tId = threadIdx.x;
	unsigned int laneId = tId & 31;
	unsigned int gId = (blockIdx.x * blockDim.x + tId);
	unsigned int slc = gId >> 5; // 5: minimum 1 WARP (2^5) 
	DTYPE tmp = 0, tmp_val;
		              	              
	if(slc < nSlices){ 	    

		unsigned int mappedSlc = dSlcMapperBin[slc];
		unsigned int idx0 = dfbrIdx0[mappedSlc] ;//slc;
    	int fb_st = fbrPtr0[mappedSlc];
		int fb_end = fbrPtr0[mappedSlc+1];

		for (int fbr = fb_st; fbr < fb_end; fbr++){
			tmp_val = 0;
	        for(unsigned int x = fbrPtr1[fbr]; x < fbrPtr1[fbr+1]; ++x) {

		        unsigned int idx2 = dInds2[x];                
	            for(unsigned int r=laneId; r<R; r+=32) {
	                tmp_val += vals[x] * dU2[idx2 * R + r]; 
	            }
	        }
	        unsigned int idx1 = fbrIdx1[fbr];
	        for(unsigned int r=laneId; r<R; r+=32) {  
	        	dU0[idx0 * R + r] += tmp_val * dU1[idx1 * R + r] ;     
	        }    
		}
	}
}

// CUDA kernel call to do HCSR MTTKRP 
__global__ void mttkrp_HCSR_kernel_smllBin(DTYPE * vals, ITYPE *dfbrIdx0, ITYPE *dSlcMapperBin, ITYPE *dInds2, ITYPE *fbrPtr0,
	ITYPE *fbrPtr1, ITYPE *fbrIdx1, unsigned int nSlices, DTYPE *dU0, DTYPE * dU1, DTYPE *dU2, 
	ITYPE	mode, ITYPE R, ITYPE warpPerSlice, int logOfWPC, int TbPerSlc, int LogOfTPS){

	unsigned int tId = threadIdx.x;
	unsigned int laneId = tId & 31;
	unsigned int gId = (blockIdx.x * blockDim.x + tId);
	unsigned int workId = (tId & ((1 << (5 + logOfWPC)) - 1)) >> 5;  
	unsigned int slc = gId >> (5 + logOfWPC); // 5: minimum 1 WARP (2^5) 
	// unsigned int slcPerTb = 16/warpPerSlice;
	// unsigned int shSlc = slc & slcPerTb;
	DTYPE tmp = 0, tmp_val;
		              	              
	if(slc < nSlices){ 	    

		unsigned int mappedSlc = dSlcMapperBin[slc];
		unsigned int idx0 = dfbrIdx0[mappedSlc] ;//slc;
    	int fb_st = fbrPtr0[mappedSlc];
		int fb_end = fbrPtr0[mappedSlc+1];

		for (int fbr = fb_st + workId; fbr < fb_end; fbr+=warpPerSlice){
			tmp_val = 0;
	        for(unsigned int x = fbrPtr1[fbr]; x < fbrPtr1[fbr+1]; ++x) {

		        unsigned int idx2 = dInds2[x];                
	            for(unsigned int r=laneId; r<R; r+=32) {
	                tmp_val += vals[x] * dU2[idx2 * R + r]; 
	            }
	        }
	        unsigned int idx1 = fbrIdx1[fbr];// dInds1[fbrPtr1[fbr]];    
	        for(unsigned int r=laneId; r<R; r+=32) {  
	        	tmp += tmp_val * dU1[idx1 * R + r] ;     
	        }    
		}

		for(unsigned int r=laneId; r<R; r+=32) {  
			atomicAdd(&dU0[idx0 * R + r], tmp);       
		}
	}
}

// CUDA kernel call to do HCSR MTTKRP 
__global__ void mttkrp_HCSR_kernel_smllBin_4D(DTYPE * vals, ITYPE *dfbrIdx0, ITYPE *dSlcMapperBin, ITYPE *dInds3, ITYPE *fbrPtr0,
	ITYPE *fbrPtr1, ITYPE *fbrIdx1, ITYPE *fbrPtr2, ITYPE *fbrIdx2, unsigned int nSlices, DTYPE *dU0, DTYPE * dU1, DTYPE *dU2, DTYPE *dU3, 
	ITYPE	mode, ITYPE R, ITYPE warpPerSlice, int logOfWPC, int TbPerSlc, int LogOfTPS){

	unsigned int tId = threadIdx.x;
	unsigned int laneId = tId & 31;
	unsigned int gId = (blockIdx.x * blockDim.x + tId);
	unsigned int workId = (tId & ((1 << (5 + logOfWPC)) - 1)) >> 5;  
	unsigned int slc = gId >> (5 + logOfWPC); // 5: minimum 1 WARP (2^5) 
	DTYPE outbuffer = 0, tmp_val = 0, outbuffer1 = 0;
		              	              
	if(slc < nSlices){ 	    

		unsigned int mappedSlc = dSlcMapperBin[slc];
		
		unsigned int idx0 = dfbrIdx0[mappedSlc] ;//slc;

		for (int fbrS = fbrPtr0[mappedSlc]; fbrS < fbrPtr0[mappedSlc+1]; fbrS++){
			
			unsigned int idx1 = fbrIdx1[fbrS];// dInds1[fbrPtr1[fbr]];  
			outbuffer1 = 0;
			
			for (int fbr = fbrPtr1[fbrS] + workId; fbr < fbrPtr1[fbrS+1]; fbr+=warpPerSlice){
				ITYPE idx2 = fbrIdx2[fbr];
				tmp_val = 0;
	    
		        for(unsigned int x = fbrPtr2[fbr]; x < fbrPtr2[fbr+1]; ++x) {

			        unsigned int idx3 = dInds3[x];                
		            for(unsigned int r=laneId; r<R; r+=32) 
		                tmp_val += vals[x] * dU3[idx3 * R + r]; 
		        }       
		        for(unsigned int r=laneId; r<R; r+=32)  
		        	outbuffer1 += tmp_val * dU2[idx2 * R + r] ;       
		    }
		    for(unsigned int r=laneId; r<R; r+=32) 
	        	outbuffer += outbuffer1 * dU1[idx1 * R + r] ;    
		}
		for(unsigned int r=laneId; r<R; r+=32) {  
			atomicAdd(&dU0[idx0 * R + r], outbuffer);  
		}
	}
}

// CUDA kernel call to do HCSR MTTKRP 
__global__ void mttkrp_HCSR_kernel_hvyBin(DTYPE * vals, ITYPE *dfbrIdx0, ITYPE *dSlcMapperBin, ITYPE *dInds2, ITYPE *fbrPtr0,
	ITYPE *fbrPtr1, ITYPE *fbrIdx1, unsigned int nSlices, DTYPE *dU0, DTYPE * dU1, DTYPE *dU2, 
	ITYPE	mode, ITYPE R, ITYPE warpPerSlice, int logOfWPC, int TbPerSlc, int logOfTPS){
	
	unsigned int laneId = threadIdx.x & 31;
	unsigned int workId = threadIdx.x >> 5;
	unsigned int slc = blockIdx.x >> logOfTPS;
	unsigned int localBId = blockIdx.x & (TbPerSlc -1);
	
	DTYPE tmp = 0, tmp_val;
		              	              
	if(slc < nSlices){

		unsigned int mappedSlc = dSlcMapperBin[slc];
		unsigned int idx0 = dfbrIdx0[mappedSlc] ;//slc;
		unsigned int nFbr = fbrPtr0[mappedSlc+1] - fbrPtr0[mappedSlc];		
		unsigned int fbrPerTb = (nFbr + TbPerSlc - 1 ) >> logOfTPS; 
		unsigned int fb_st = fbrPtr0[mappedSlc] + localBId * fbrPerTb ;
		unsigned int fb_end = fbrPtr0[mappedSlc] + (localBId + 1) * fbrPerTb ;

		for (int fbr = fb_st + workId; fbr < fb_end && fbr < fbrPtr0[mappedSlc+1] ; fbr+=warpPerSlice){
			tmp_val = 0;
	        
	        for(unsigned int x = fbrPtr1[fbr]; x < fbrPtr1[fbr+1]; ++x) {

		        unsigned int idx2 = dInds2[x];                
	            for(unsigned int r=laneId; r<R; r+=32) {
	                tmp_val += vals[x] * dU2[idx2 * R + r]; 
	            }
	        }
	        unsigned int idx1 = fbrIdx1[fbr];//dInds1[fbrPtr1[fbr]];    
	        for(unsigned int r=laneId; r<R; r+=32) {  
	        	tmp += tmp_val * dU1[idx1 * R + r] ;     
	            // // atomicAdd(&dU0[idx0 * R + r], tmp);
	        }    
		}
		for(unsigned int r=laneId; r<R; r+=32) {  
            atomicAdd(&dU0[idx0 * R + r], tmp);
        } 
	}
}

// CUDA kernel call to do HCSR MTTKRP 
__global__ void mttkrp_HCSR_kernel_hvyBin_4D(DTYPE * vals, ITYPE *dfbrIdx0, ITYPE *dSlcMapperBin, ITYPE *dInds3, ITYPE *fbrPtr0,
	ITYPE *fbrPtr1, ITYPE *fbrIdx1, ITYPE *fbrPtr2, ITYPE *fbrIdx2, unsigned int nSlices, DTYPE *dU0, DTYPE * dU1, DTYPE *dU2, DTYPE *dU3, 
	ITYPE	mode, ITYPE R, ITYPE warpPerSlice, int logOfWPC, int TbPerSlc, int logOfTPS){
	
	unsigned int laneId = threadIdx.x & 31;
	unsigned int workId = threadIdx.x >> 5;
	unsigned int slc = blockIdx.x >> logOfTPS;
	unsigned int localBId = blockIdx.x & (TbPerSlc -1);
	
	DTYPE outbuffer = 0, tmp_val = 0, outbuffer1 = 0;;
		              	              
	if(slc < nSlices){

		unsigned int mappedSlc = dSlcMapperBin[slc];
		unsigned int idx0 = dfbrIdx0[mappedSlc] ;//slc;
		unsigned int nFbr = fbrPtr0[mappedSlc+1] - fbrPtr0[mappedSlc];		
		unsigned int fbrPerTb = (nFbr + TbPerSlc - 1 ) >> logOfTPS; 
		unsigned int fb_st = fbrPtr0[mappedSlc] + localBId * fbrPerTb ;
		unsigned int fb_end = fbrPtr0[mappedSlc] + (localBId + 1) * fbrPerTb ;

		for (int fbrS = fb_st; fbrS < fb_end && fbrS < fbrPtr0[mappedSlc+1] ; fbrS++){
			unsigned int idx1 = fbrIdx1[fbrS];// dInds1[fbrPtr1[fbr]];  
			outbuffer1 = 0;

			for (int fbr = fbrPtr1[fbrS] + workId; fbr < fbrPtr1[fbrS+1]; fbr+=warpPerSlice){
				ITYPE idx2 = fbrIdx2[fbr];
				tmp_val = 0;
            
		        for(unsigned int x = fbrPtr2[fbr]; x < fbrPtr2[fbr+1]; ++x) {

			        unsigned int idx3 = dInds3[x];                
		            for(unsigned int r=laneId; r<R; r+=32) 
		                tmp_val += vals[x] * dU3[idx3 * R + r]; 
		        }
		        for(unsigned int r=laneId; r<R; r+=32)  
		        	outbuffer1 += tmp_val * dU2[idx2 * R + r] ;  
		    }
		    for(unsigned int r=laneId; r<R; r+=32) 
	        	outbuffer += outbuffer1 * dU1[idx1 * R + r] ;     
		}
		for(unsigned int r=laneId; r<R; r+=32) { 
            atomicAdd(&dU0[idx0 * R + r], outbuffer);
        } 
	}
}

// CUDA fbr atomic sing slcLikeFbr
__global__ void mttkrp_MIHCSR_kernel_slc_atomic_fbrLvlPar(DTYPE * vals, ITYPE *fbrLikeSlcInds, ITYPE *dInds2, 
	ITYPE *fbrPtr0, ITYPE *fbrPtr1, ITYPE *fbrIdx1, unsigned int nFibers, DTYPE *dU0, DTYPE * dU1, DTYPE *dU2, 
	ITYPE	mode, ITYPE R, ITYPE warpPerSlice, int logOfWPC, int fbrPerWarp, int logOfFPW){

	unsigned int tId = threadIdx.x;
	unsigned int laneId = tId & 31;
	unsigned int gId = (blockIdx.x * blockDim.x + tId);
	unsigned int workId = (tId & ((1 << (5 + logOfWPC)) - 1)) >> 5;  //tId >> 5; //tId >> 5;//
	unsigned int fbr = (gId >> (5 + logOfWPC)) << logOfFPW; // 5: minimum 1 WARP (2^5) // blockIdx.x ;//
	DTYPE tmp = 0, tmp_val;
		              	              
	if(fbr < nFibers - 1){ 	    
		
		tmp_val = 0;
		bool diffFiber = false;
		unsigned int idx0;

		for (int fr = 0; fr < fbrPerWarp && (fbr+fr) < (nFibers - 1); ++fr){

			diffFiber = false;
			unsigned int idx1 = fbrIdx1[fbr+fr];// dInds1[fbrPtr1[fbr]];  
			idx0 = fbrLikeSlcInds[fbr+fr];//slc;  
 			tmp_val = 0;
	        for(unsigned int x = fbrPtr1[fbr+fr] + workId; x < fbrPtr1[fbr+fr+1]; x+=warpPerSlice) {

		        unsigned int idx2 = dInds2[x];                    

	            for(unsigned int r=laneId; r<R; r+=32) {
	                tmp_val += vals[x] * dU2[idx2 * R + r]; //2MR   
	            }       
	        }
	        	
        	for(unsigned int r=laneId; r<R; r+=32) { 
        		tmp += tmp_val * dU1[idx1 * R + r] ;
        	} 
	        
        	if(fbrLikeSlcInds[fbr+fr] != fbrLikeSlcInds[fbr+fr+1]) {

        		diffFiber = true;
	        	for(unsigned int r=laneId; r<R; r+=32) { 
	        		atomicAdd(&dU0[idx0 * R + r], tmp); //2PR
	        	} 
        		tmp = 0;
        	}
        } 

        if(!diffFiber) {  
	        for(unsigned int r=laneId; r<R; r+=32) { 
	        	atomicAdd(&dU0[idx0 * R + r], tmp); //2PR
	        }  
        }  
	}
}

// CUDA kernel call to do HCSR MTTKRP 
__global__ void mttkrp_MIHCSR_kernel_smllBin_fbr_atomic(DTYPE * vals, ITYPE *dfbrIdx0, ITYPE *dSlcMapperBin, ITYPE *dInds2, ITYPE *fbrPtr0,
	ITYPE *fbrPtr1, ITYPE *fbrIdx1, unsigned int nSlices, DTYPE *dU0, DTYPE * dU1, DTYPE *dU2, 
	ITYPE	mode, ITYPE R, ITYPE warpPerSlice, int logOfWPC, int TbPerSlc, int LogOfTPS){

	unsigned int tId = threadIdx.x;
	unsigned int laneId = tId & 31;
	unsigned int gId = (blockIdx.x * blockDim.x + tId);
	unsigned int workId = (tId & ((1 << (5 + logOfWPC)) - 1)) >> 5;  
	unsigned int slc = gId >> (5 + logOfWPC); // 5: minimum 1 WARP (2^5) 
	DTYPE tmp = 0, tmp_val;
		              	              
	if(slc < nSlices){ 	    

		unsigned int mappedSlc = dSlcMapperBin[slc];
		unsigned int idx2 = dfbrIdx0[mappedSlc] ;//slc;
    	int fb_st = fbrPtr0[mappedSlc];
		int fb_end = fbrPtr0[mappedSlc+1];

		for (int fbr = fb_st + workId; fbr < fb_end; fbr+=warpPerSlice){
			
			tmp_val = 0;
			unsigned int idx0 = fbrIdx1[fbr];// dInds1[fbrPtr1[fbr]];    
	        
	        for(unsigned int x = fbrPtr1[fbr]; x < fbrPtr1[fbr+1]; ++x) {

		        unsigned int idx1 = dInds2[x];                    

	            for(unsigned int r=laneId; r<R; r+=32) {
	                tmp_val += vals[x] * dU1[idx1 * R + r]; //2MR
	            }
	        }     
	        for(unsigned int r=laneId; r<R; r+=32) { 
	        	tmp = tmp_val * dU2[idx2 * R + r] ;
	        	atomicAdd(&dU0[idx0 * R + r], tmp); //2PR
	        }    
		}
	}
}

// CUDA fbr atomic sing slcLikeFbr
__global__ void mttkrp_MIHCSR_kernel_fbr_atomic_fbrLvlPar(DTYPE * vals, ITYPE *fbrLikeSlcInds, ITYPE *dInds2, 
	ITYPE *fbrPtr0, ITYPE *fbrPtr1, ITYPE *fbrIdx1, unsigned int nFibers, DTYPE *dU0, DTYPE * dU1, DTYPE *dU2, 
	ITYPE	mode, ITYPE R, ITYPE warpPerSlice, int logOfWPC){

	unsigned int tId = threadIdx.x;
	unsigned int laneId = tId & 31;
	unsigned int gId = (blockIdx.x * blockDim.x + tId);
	unsigned int workId = (tId & ((1 << (5 + logOfWPC)) - 1)) >> 5;  //tId >> 5; //tId >> 5;//
	unsigned int fbr = gId >> (5 + logOfWPC); // 5: minimum 1 WARP (2^5) // blockIdx.x ;//
	DTYPE tmp = 0, tmp_val;
		              	              
	if(fbr < nFibers - 1){ 	    
		
		tmp_val = 0;
		unsigned int idx0 = fbrIdx1[fbr];// dInds1[fbrPtr1[fbr]];  
		unsigned int idx2 = fbrLikeSlcInds[fbr];//slc;  
        
        for(unsigned int x = fbrPtr1[fbr] + workId; x < fbrPtr1[fbr+1]; x+=warpPerSlice) {

	        unsigned int idx1 = dInds2[x];                    

            for(unsigned int r=laneId; r<R; r+=32) {
                tmp_val += vals[x] * dU1[idx1 * R + r]; //2MR
            }
        }     
        for(unsigned int r=laneId; r<R; r+=32) { 
        	tmp = tmp_val * dU2[idx2 * R + r] ;
        	atomicAdd(&dU0[idx0 * R + r], tmp); //2PR
        }    
	}
}

// CUDA fbr atomic sing slcLikeFbr
__global__ void mttkrp_MIHCSR_kernel_fbr_atomic_fbrLvlPar_loop(DTYPE * vals, ITYPE *fbrLikeSlcInds, ITYPE *dInds2, 
	ITYPE *fbrPtr0, ITYPE *fbrPtr1, ITYPE *fbrIdx1, unsigned int nFibers, DTYPE *dU0, DTYPE * dU1, DTYPE *dU2, 
	ITYPE	mode, ITYPE R, ITYPE warpPerSlice, int logOfWPC){

	unsigned int tId = threadIdx.x;
	unsigned int laneId = tId & 31;
	unsigned int gId = (blockIdx.x * blockDim.x + tId);

	//like PARTI
	//hardcoded for 1 warp per nnz
	size_t num_loops_fbr = 1 * 32;
    size_t const fbr_per_loop = gridDim.x * blockDim.x;
    if(nFibers > fbr_per_loop) {
        num_loops_fbr = ((nFibers + fbr_per_loop - 1) / fbr_per_loop) << 5;
    }

	DTYPE tmp = 0, tmp_val;

	unsigned int fbr;

	for(size_t nl=0; nl<num_loops_fbr; ++nl) {
		
		fbr = (gId + nl * fbr_per_loop) >> 5;
		              	              
		if(fbr < nFibers - 1){ 	    
			
			tmp_val = 0;
			unsigned int idx0 = fbrIdx1[fbr];// dInds1[fbrPtr1[fbr]];  
			unsigned int idx2 = fbrLikeSlcInds[fbr];//slc;  
	        
	        for(unsigned int x = fbrPtr1[fbr]; x < fbrPtr1[fbr+1]; x++) {

		        unsigned int idx1 = dInds2[x];                    

	            for(unsigned int r=laneId; r<R; r+=32) {
	                tmp_val += vals[x] * dU1[idx1 * R + r]; //2MR
	            }
	        }     
	        for(unsigned int r=laneId; r<R; r+=32) { 
	        	tmp = tmp_val * dU2[idx2 * R + r] ;
	        	atomicAdd(&dU0[idx0 * R + r], tmp); //2PR
	        }    
		}
	}
}

// CUDA kernel call to do HCSR MTTKRP 
__global__ void mttkrp_MIHCSR_kernel_hvyBin_fbr_atomic(DTYPE * vals, ITYPE *dfbrIdx0, ITYPE *dSlcMapperBin, ITYPE *dInds2, ITYPE *fbrPtr0,
	ITYPE *fbrPtr1, ITYPE *fbrIdx1, unsigned int nSlices, DTYPE *dU0, DTYPE * dU1, DTYPE *dU2, 
	ITYPE	mode, ITYPE R, ITYPE warpPerSlice, int logOfWPC, int TbPerSlc, int logOfTPS){
	
	unsigned int laneId = threadIdx.x & 31;
	unsigned int workId = threadIdx.x >> 5;
	unsigned int slc = blockIdx.x >> logOfTPS;
	unsigned int localBId = blockIdx.x & (TbPerSlc -1);
	
	DTYPE tmp = 0, tmp_val;
		              	              
	if(slc < nSlices){

		unsigned int mappedSlc = dSlcMapperBin[slc];
		unsigned int idx2 = dfbrIdx0[mappedSlc] ;//slc;
		unsigned int nFbr = fbrPtr0[mappedSlc+1] - fbrPtr0[mappedSlc];		
		unsigned int fbrPerTb = (nFbr + TbPerSlc - 1 ) >> logOfTPS; 
		unsigned int fb_st = fbrPtr0[mappedSlc] + localBId * fbrPerTb ;
		unsigned int fb_end = fbrPtr0[mappedSlc] + (localBId + 1) * fbrPerTb ;

		for (int fbr = fb_st + workId; fbr < fb_end && fbr < fbrPtr0[mappedSlc+1]; fbr+=warpPerSlice){
			
			tmp_val = 0;
			unsigned int idx0 = fbrIdx1[fbr];// dInds1[fbrPtr1[fbr]];    
	        
	        for(unsigned int x = fbrPtr1[fbr]; x < fbrPtr1[fbr+1]; ++x) {

		        unsigned int idx1 = dInds2[x];                    

	            for(unsigned int r=laneId; r<R; r+=32) {
	                tmp_val += vals[x] * dU1[idx1 * R + r]; 
	            }
	        }     
	        for(unsigned int r=laneId; r<R; r+=32) { 
	        	tmp = tmp_val * dU2[idx2 * R + r] ;
	        	atomicAdd(&dU0[idx0 * R + r], tmp); 
	        }    
		} 
	}
}

// CUDA kernel call to do HCSR MTTKRP 
__global__ void mttkrp_MIHCSR_kernel_smllBin_all_atomic(DTYPE * vals, ITYPE *dfbrIdx0, ITYPE *dSlcMapperBin, ITYPE *dInds2, ITYPE *fbrPtr0,
	ITYPE *fbrPtr1, ITYPE *fbrIdx1, unsigned int nSlices, DTYPE *dU0, DTYPE * dU1, DTYPE *dU2, 
	ITYPE	mode, ITYPE R, ITYPE warpPerSlice, int logOfWPC, int TbPerSlc, int LogOfTPS){

	unsigned int tId = threadIdx.x;
	unsigned int laneId = tId & 31;
	unsigned int gId = (blockIdx.x * blockDim.x + tId);
	unsigned int workId = (tId & ((1 << (5 + logOfWPC)) - 1)) >> 5;  
	unsigned int slc = gId >> (5 + logOfWPC); // 5: minimum 1 WARP (2^5) 
	// unsigned int slcPerTb = 16/warpPerSlice;
	// unsigned int shSlc = slc & slcPerTb;
	DTYPE tmp_val;
		              	              
	if(slc < nSlices){ 	    

		unsigned int mappedSlc = dSlcMapperBin[slc];
		unsigned int idx1 = dfbrIdx0[mappedSlc] ;//slc;
    	int fb_st = fbrPtr0[mappedSlc];
		int fb_end = fbrPtr0[mappedSlc+1];

		for (int fbr = fb_st + workId; fbr < fb_end; fbr+=warpPerSlice){
			
			unsigned int idx2 = fbrIdx1[fbr];// dInds1[fbrPtr1[fbr]];  

            // for(unsigned int r=laneId; r<R; r+=32) 
            // 	tmp_val = dU1[idx1 * R + r] * dU2[idx2 * R + r] ; //1PR
	        
	        for(unsigned int x = fbrPtr1[fbr]; x < fbrPtr1[fbr+1]; ++x) {

		        unsigned int idx0 = dInds2[x];                    

	            for(unsigned int r=laneId; r<R; r+=32) {
	            	tmp_val =  vals[x] * dU1[idx1 * R + r] * dU2[idx2 * R + r] ;
	            	atomicAdd(&dU0[idx0 * R + r], tmp_val); //2MR
	                // atomicAdd(&dU0[idx0 * R + r], (tmp_val * vals[x]) ); 
	            }
	        }   	
		}
	}
}

// CUDA fbr atomic sing slcLikeFbr
__global__ void mttkrp_MIHCSR_kernel_all_atomic_fbrLvlPar(DTYPE * vals, ITYPE *fbrLikeSlcInds, ITYPE *dInds2, 
	ITYPE *fbrPtr0, ITYPE *fbrPtr1, ITYPE *fbrIdx1, unsigned int nFibers, DTYPE *dU0, DTYPE * dU1, DTYPE *dU2, 
	ITYPE	mode, ITYPE R, ITYPE warpPerSlice, int logOfWPC){

	unsigned int tId = threadIdx.x;
	unsigned int laneId = tId & 31;
	unsigned int gId = (blockIdx.x * blockDim.x + tId);
	unsigned int workId = (tId & ((1 << (5 + logOfWPC)) - 1)) >> 5;  //tId >> 5; //tId >> 5;//
	unsigned int fbr = gId >> (5 + logOfWPC); // 5: minimum 1 WARP (2^5) // blockIdx.x ;//
	DTYPE tmp = 0, tmp_val;
		              	              
	if(fbr < nFibers - 1){ 	    
		
		tmp_val = 0;
		unsigned int idx2 = fbrIdx1[fbr];// dInds1[fbrPtr1[fbr]];  
		unsigned int idx1 = fbrLikeSlcInds[fbr];//slc;  

		for(unsigned int r=laneId; r<R; r+=32) 
           	tmp = dU1[idx1 * R + r] * dU2[idx2 * R + r] ; //1PR
        
        for(unsigned int x = fbrPtr1[fbr] + workId; x < fbrPtr1[fbr+1]; x+=warpPerSlice) {

	        unsigned int idx0 = dInds2[x];                    

            for(unsigned int r=laneId; r<R; r+=32) {
                tmp_val = vals[x] * tmp;///dU1[idx1 * R + r] * dU2[idx2 * R + r] ; //2MR
                atomicAdd(&dU0[idx0 * R + r], tmp_val);
            }
        }         
	}
}

// CUDA fbr atomic sing slcLikeFbr
__global__ void mttkrp_MIHCSR_kernel_all_atomic_fbrLvlPar_loop(DTYPE * vals, ITYPE *fbrLikeSlcInds, ITYPE *dInds2, 
	ITYPE *fbrPtr0, ITYPE *fbrPtr1, ITYPE *fbrIdx1, unsigned int nFibers, DTYPE *dU0, DTYPE * dU1, DTYPE *dU2, 
	ITYPE	mode, ITYPE R, ITYPE warpPerSlice, int logOfWPC){

	unsigned int tId = threadIdx.x;
	unsigned int laneId = tId & 31;
	unsigned int gId = (blockIdx.x * blockDim.x + tId);
	unsigned int warpId = (tId & ((1 << (5 + logOfWPC)) - 1)) >> 5;  //tId >> 5; //
	unsigned int blockId = gId >> (5 + logOfWPC); // 5: minimum 1 WARP (2^5) //blockIdx.x ;//

	//like PARTI
	//hardcoded for 1 warp per nnz
	size_t num_loops_fbr = 1 * 32;
    size_t const fbr_per_loop = gridDim.x * blockDim.x;
    if(nFibers > fbr_per_loop) {
        num_loops_fbr = ((nFibers + fbr_per_loop - 1) / fbr_per_loop) << 5;
    }

	DTYPE tmp = 0, tmp_val;

	unsigned int fbr;

	for(size_t nl=0; nl<num_loops_fbr; ++nl) {
		
		fbr = (gId + nl * fbr_per_loop) >> 5;
		              	              
		if(fbr < nFibers - 1){ 	    
			
			tmp_val = 0;
			unsigned int idx2 = fbrIdx1[fbr];// dInds1[fbrPtr1[fbr]];  
			unsigned int idx1 = fbrLikeSlcInds[fbr];//slc;  

			for(unsigned int r=laneId; r<R; r+=32) 
	           	tmp = dU1[idx1 * R + r] * dU2[idx2 * R + r] ; //1PR
	        
	        for(unsigned int x = fbrPtr1[fbr] + warpId; x < fbrPtr1[fbr+1]; x+=warpPerSlice) {

		        unsigned int idx0 = dInds2[x];                    

	            for(unsigned int r=laneId; r<R; r+=32) {
	                tmp_val = vals[x] * tmp;///dU1[idx1 * R + r] * dU2[idx2 * R + r] ; //2MR
	                atomicAdd(&dU0[idx0 * R + r], tmp_val);
	            }
	        }    
		}
	}
}

// CUDA kernel call to do HCSR MTTKRP 
__global__ void mttkrp_MIHCSR_kernel_hvyBin_all_atomic(DTYPE * vals, ITYPE *dfbrIdx0, ITYPE *dSlcMapperBin, ITYPE *dInds2, ITYPE *fbrPtr0,
	ITYPE *fbrPtr1, ITYPE *fbrIdx1, unsigned int nSlices, DTYPE *dU0, DTYPE * dU1, DTYPE *dU2, 
	ITYPE	mode, ITYPE R, ITYPE warpPerSlice, int logOfWPC, int TbPerSlc, int logOfTPS){
	
	unsigned int laneId = threadIdx.x & 31;
	unsigned int workId = threadIdx.x >> 5;
	unsigned int slc = blockIdx.x >> logOfTPS;
	unsigned int localBId = blockIdx.x & (TbPerSlc -1);
	
	DTYPE tmp = 0, tmp_val;
		              	              
	if(slc < nSlices){

		unsigned int mappedSlc = dSlcMapperBin[slc];
		unsigned int idx1 = dfbrIdx0[mappedSlc] ;//slc;
		unsigned int nFbr = fbrPtr0[mappedSlc+1] - fbrPtr0[mappedSlc];		
		unsigned int fbrPerTb = (nFbr + TbPerSlc - 1 ) >> logOfTPS; 
		unsigned int fb_st = fbrPtr0[mappedSlc] + localBId * fbrPerTb ;
		unsigned int fb_end = fbrPtr0[mappedSlc] + (localBId + 1) * fbrPerTb ;

		for (int fbr = fb_st + workId; fbr < fb_end && fbr < fbrPtr0[mappedSlc+1]; fbr+=warpPerSlice){
			
			tmp_val = 0;
			unsigned int idx2 = fbrIdx1[fbr];// dInds1[fbrPtr1[fbr]]; 

			for(unsigned int r=laneId; r<R; r+=32) 
            	tmp_val = dU1[idx1 * R + r] * dU2[idx2 * R + r] ;  
	        
	        for(unsigned int x = fbrPtr1[fbr]; x < fbrPtr1[fbr+1]; ++x) {

		        unsigned int idx0 = dInds2[x];                    

	            for(unsigned int r=laneId; r<R; r+=32) {
	            	// atomicAdd(&dU0[idx0 * R + r], (tmp_val * vals[x]) ); 
	            	tmp_val =  vals[x] * dU1[idx1 * R + r] * dU2[idx2 * R + r] ;
	                atomicAdd(&dU0[idx0 * R + r], tmp_val); 
	            }
	        }    
		} 
	}
}


int MTTKRP_COO_GPU(const Tensor &X, Matrix *U, const Options Opt){
	//allocate and memcpy GPU memory

	//Tensor
	ITYPE mode = Opt.mode;
	ITYPE R = Opt.R;
	ITYPE *dInds0, *dInds1, *dInds2, *dInds3;
	DTYPE *dVals;

	ITYPE mode0 = X.modeOrder[0];
    ITYPE mode1 = X.modeOrder[1];
    ITYPE mode2 = X.modeOrder[2];

	checkCuda(hipMalloc((void**) &dVals, X.totNnz * sizeof(DTYPE)), 0);
	checkCuda(hipMalloc((void**) &dInds0, X.totNnz * sizeof(ITYPE)), 0);
	checkCuda(hipMalloc((void**) &dInds1, X.totNnz * sizeof(ITYPE)), 0);
	checkCuda(hipMalloc((void**) &dInds2, X.totNnz * sizeof(ITYPE)), 0);

	checkCuda(hipMemcpy(dVals, &(X.vals[0]), X.totNnz * sizeof(DTYPE),hipMemcpyHostToDevice), 0);
	checkCuda(hipMemcpy(dInds0, &(X.inds[mode0][0]), X.totNnz * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
	checkCuda(hipMemcpy(dInds1, &(X.inds[mode1][0]), X.totNnz * sizeof(ITYPE) ,hipMemcpyHostToDevice), 0);
	checkCuda(hipMemcpy(dInds2, &(X.inds[mode2][0]), X.totNnz * sizeof(ITYPE),hipMemcpyHostToDevice), 0);

	// //Matrices
	DTYPE *dU0, *dU1, *dU2, *dU3;	
	checkCuda(hipMalloc((void**) &dU0, U[mode0].nRows * U[mode0].nCols * sizeof(DTYPE)), 0);
	checkCuda(hipMalloc((void**) &dU1, U[mode1].nRows * U[mode1].nCols * sizeof(DTYPE)), 0);
	checkCuda(hipMalloc((void**) &dU2, U[mode2].nRows * U[mode2].nCols * sizeof(DTYPE)), 0);

	hipMemset(dU0, 0,  U[mode0].nRows * U[mode0].nCols * sizeof(DTYPE));
	checkCuda(hipMemcpy(dU1, &(U[mode1].vals[0]), U[mode1].nRows * U[mode1].nCols * sizeof(DTYPE), hipMemcpyHostToDevice), 0);
	checkCuda(hipMemcpy(dU2, &(U[mode2].vals[0]), U[mode2].nRows * U[mode2].nCols * sizeof(DTYPE), hipMemcpyHostToDevice), 0);

	if(X.ndims == 4){
		ITYPE mode3 = X.modeOrder[3];
		checkCuda(hipMalloc((void**) &dInds3, X.totNnz * sizeof(ITYPE)), 0);
		checkCuda(hipMemcpy(dInds3, &(X.inds[mode3][0]), X.totNnz * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
		checkCuda(hipMalloc((void**) &dU3, U[mode3].nRows * U[mode3].nCols * sizeof(DTYPE)), 0);
		checkCuda(hipMemcpy(dU3, &(U[mode3].vals[0]), U[mode3].nRows * U[mode3].nCols * sizeof(DTYPE), hipMemcpyHostToDevice), 0);
	}
	
	// BLOCK and GRID
	int BLOCKSIZE = 128;
	dim3 block(BLOCKSIZE, 1, 1), grid(1, 1, 1);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float mili = 0;
    bool useLoop = true;
	
	// /* Like PARTI loop */ = 
	if(useLoop)
		grid.x = 32768;
	else 
		grid.x = (32 * X.totNnz + BLOCKSIZE - 1) / BLOCKSIZE;
	
	// CUDA call
	cuda_timer_start(start);

	if(!useLoop){

		if(X.ndims == 3)
			mttkrp_COO_kernel<<<grid, block>>>(dVals, dInds0, dInds1, dInds2, X.totNnz, dU0, dU1, dU2, mode, R); 
		
		else if(X.ndims == 4)
			mttkrp_COO_kernel_4D<<<grid, block>>>(dVals, dInds0, dInds1, dInds2, dInds3, X.totNnz, dU0, dU1, dU2, dU3, mode, R); 
	
	}
	// /* loop like ParTI */
	else{

		if(X.ndims == 3)
			mttkrp_COO_kernel_loop<<<grid, block>>>(dVals, dInds0, dInds1, dInds2, X.totNnz, dU0, dU1, dU2, mode, R ); 
		
		else if(X.ndims == 4)
			mttkrp_COO_kernel_4D_loop<<<grid, block>>>(dVals, dInds0, dInds1, dInds2, dInds3, X.totNnz, dU0, dU1, dU2, dU3, mode, R); 
	
	}
	cuda_timer_stop(start, stop, mili);

	if(useLoop) cout << "Loop on. ";
    cout << "COO GPU using loop - time " << mili << "ms"<< endl;

	// check correctness
	checkCuda(hipMemcpy(&U[mode0].vals[0], dU0, U[mode0].nRows * U[mode0].nCols * sizeof(DTYPE), hipMemcpyDeviceToHost), 0);
	// print_output(U, 0);
	hipFree(dVals); 
	hipFree(dU0); hipFree(dU1); hipFree(dU2); hipFree(dU3);
	hipFree(dInds0); hipFree(dInds1); hipFree(dInds2); hipFree(dInds3);


	return 0;
}

int MTTKRP_HCSR_GPU(Tensor &X, Matrix *U, const Options &Opt){
	//allocate and memcpy GPU memory
	cout << "FIX fiber idx" << endl;
	//Tensor
	ITYPE *dInds2, *dInds3, *dfbrPtr0, *dfbrIdx0, *dfbrPtr1, *dfbrIdx1, *dFbrPtr2, *dFbrIdx2, *dSlcMapperBin;
	DTYPE *dVals;
	int logOfWarpPerSlice = log2(Opt.warpPerSlice);
	int TbPerSlc = 1;
	int logOfTPS = log2(TbPerSlc);

	ITYPE mode0 = X.modeOrder[0];
    ITYPE mode1 = X.modeOrder[1];
    ITYPE mode2 = X.modeOrder[2];

	// dummy bin mapper to be compatible with bin mapper when bin are not used
	X.slcMapperBin.push_back(std::vector<ITYPE>());      
	for (int s = 0; s < X.fbrIdx[0].size(); ++s)
		X.slcMapperBin[0].push_back(s);

	checkCuda(hipMalloc((void**) &dVals, X.totNnz * sizeof(DTYPE)), 0);
	checkCuda(hipMalloc((void**) &dSlcMapperBin, X.slcMapperBin[0].size() * sizeof(ITYPE)), 0);
	checkCuda(hipMalloc((void**) &dfbrIdx0, X.fbrIdx[0].size() * sizeof(ITYPE)), 0);
	checkCuda(hipMalloc((void**) &dfbrPtr0, X.fbrPtr[0].size() * sizeof(ITYPE)), 0);
	checkCuda(hipMalloc((void**) &dfbrPtr1, X.fbrPtr[1].size() * sizeof(ITYPE)), 0);
	checkCuda(hipMalloc((void**) &dfbrIdx1, X.fbrIdx[1].size() * sizeof(ITYPE)), 0);

	checkCuda(hipMemcpy(dVals, &(X.vals[0]), X.totNnz * sizeof(DTYPE),hipMemcpyHostToDevice), 0);
	checkCuda(hipMemcpy(dSlcMapperBin, &(X.slcMapperBin[0][0]), X.slcMapperBin[0].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
	checkCuda(hipMemcpy(dfbrPtr0, &(X.fbrPtr[0][0]), X.fbrPtr[0].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
	checkCuda(hipMemcpy(dfbrIdx0, &(X.fbrIdx[0][0]), X.fbrIdx[0].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
	checkCuda(hipMemcpy(dfbrPtr1, &(X.fbrPtr[1][0]), X.fbrPtr[1].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
	checkCuda(hipMemcpy(dfbrIdx1, &(X.fbrIdx[1][0]), X.fbrIdx[1].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);

	// //Matrices
	DTYPE *dU0, *dU1, *dU2, *dU3;	
	checkCuda(hipMalloc((void**) &dU0, U[mode0].nRows * U[mode0].nCols * sizeof(DTYPE)), 0);
	checkCuda(hipMalloc((void**) &dU1, U[mode1].nRows * U[mode1].nCols * sizeof(DTYPE)), 0);
	checkCuda(hipMalloc((void**) &dU2, U[mode2].nRows * U[mode2].nCols * sizeof(DTYPE)), 0);

	hipMemset(dU0, 0,  U[mode0].nRows * U[mode0].nCols * sizeof(DTYPE));
	checkCuda(hipMemcpy(dU1, &(U[mode1].vals[0]), U[mode1].nRows * U[mode1].nCols * sizeof(DTYPE), hipMemcpyHostToDevice), 0);
	checkCuda(hipMemcpy(dU2, &(U[mode2].vals[0]), U[mode2].nRows * U[mode2].nCols * sizeof(DTYPE), hipMemcpyHostToDevice), 0);
	
	if(X.ndims == 3){
		checkCuda(hipMalloc((void**) &dInds2, X.totNnz * sizeof(ITYPE)), 0);
		checkCuda(hipMemcpy(dInds2, &(X.inds[mode2][0]), X.totNnz * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
	}

	if(X.ndims == 4){
		ITYPE mode3 = X.modeOrder[3];
		checkCuda(hipMalloc((void**) &dFbrIdx2, X.fbrIdx[2].size() * sizeof(ITYPE)), 0);
		checkCuda(hipMalloc((void**) &dFbrPtr2, X.fbrPtr[2].size() * sizeof(ITYPE)), 0);
		checkCuda(hipMalloc((void**) &dInds3, X.totNnz * sizeof(ITYPE)), 0);
		checkCuda(hipMalloc((void**) &dU3, U[mode3].nRows * U[mode3].nCols * sizeof(DTYPE)), 0);
		
		checkCuda(hipMemcpy(dFbrPtr2, &(X.fbrPtr[2][0]), X.fbrPtr[2].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
		checkCuda(hipMemcpy(dFbrIdx2, &(X.fbrIdx[2][0]), X.fbrIdx[2].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
		checkCuda(hipMemcpy(dInds3, &(X.inds[mode3][0]), X.totNnz * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
		checkCuda(hipMemcpy(dU3, &(U[mode3].vals[0]), U[mode3].nRows * U[mode3].nCols * sizeof(DTYPE), hipMemcpyHostToDevice), 0);
	}

	// BLOCK and GRID
	int BLOCKSIZE = 512;

	if(Opt.warpPerSlice * 32 > BLOCKSIZE){
		cout << "BLOCKSIZE is smaller than work per slice! Increase BLOCKSIZE." << endl;
		exit(0);
	}

	dim3 block(BLOCKSIZE, 1, 1), grid(1, 1, 1);
	grid.x = (Opt.warpPerSlice * 32 * X.dims[mode0] + BLOCKSIZE - 1) / BLOCKSIZE;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float mili = 0;

	checkCuda(hipEventRecord(start), __LINE__);

	// mttkrp_HCSR_kernel_COO<<<grid, block, 32 * sizeof(DTYPE)>>>(dVals, dfbrIdx0, dSlcMapperBin, dInds2, dfbrPtr0, dfbrPtr1, dfbrIdx1,
	// 	X.fbrIdx[0].size(), dU0, dU1, dU2,Opt.mode, Opt.R, Opt.warpPerSlice, logOfWarpPerSlice, TbPerSlc, logOfTPS); 
	if(X.ndims == 3)
		mttkrp_HCSR_kernel_smllBin<<<grid, block, 32 * sizeof(DTYPE)>>>(dVals, dfbrIdx0, dSlcMapperBin, dInds2, dfbrPtr0, dfbrPtr1, dfbrIdx1,
		X.fbrIdx[0].size(), dU0, dU1, dU2,Opt.mode, Opt.R, Opt.warpPerSlice, logOfWarpPerSlice, TbPerSlc, logOfTPS); 
	else
		mttkrp_HCSR_kernel_smllBin_4D<<<grid, block, 32 * sizeof(DTYPE)>>>(dVals, dfbrIdx0, dSlcMapperBin, dInds3, dfbrPtr0, dfbrPtr1, dfbrIdx1,
		dFbrPtr2, dFbrIdx2, X.fbrIdx[0].size(), dU0, dU1, dU2, dU3, Opt.mode, Opt.R, Opt.warpPerSlice, logOfWarpPerSlice, TbPerSlc, logOfTPS); 


	checkCuda(hipEventRecord(stop), __LINE__);
    hipEventSynchronize(stop);
    checkCuda(hipEventElapsedTime(&mili, start, stop), __LINE__);
    hipDeviceSynchronize();
    cout << "HCSR GPU - time " << mili << "ms"<< endl;

	// check correctness
	checkCuda(hipMemcpy(&U[mode0].vals[0], dU0, U[mode0].nRows * U[mode0].nCols * sizeof(DTYPE), hipMemcpyDeviceToHost), 0);
	hipFree(dVals); 
	hipFree(dU0); hipFree(dU1); hipFree(dU2); hipFree(dU3);
	hipFree(dInds2); hipFree(dInds3); 
	hipFree(dfbrIdx0); hipFree(dfbrIdx1); hipFree(dFbrIdx2);
	hipFree(dfbrPtr0); hipFree(dfbrPtr1); hipFree(dFbrPtr2);

	return 0;
}

int MTTKRP_TILED_COO_GPU(TiledTensor *TiledX, Matrix *U, const Options Opt){
	//allocate and memcpy GPU memory

	//Tensor
	ITYPE mode = Opt.mode;
	ITYPE R = Opt.R;
	ITYPE *dInds0, *dInds1, *dInds2;
	ITYPE dLoc = 0, totNnz = 0;
	DTYPE *dVals;

	// All tile same mode
	ITYPE mode0 = TiledX[0].modeOrder[0];
    ITYPE mode1 = TiledX[0].modeOrder[1];
    ITYPE mode2 = TiledX[0].modeOrder[2];

	for (int tile = 0; tile < Opt.nTile; ++tile)
		totNnz += TiledX[tile].totNnz;

	checkCuda(hipMalloc((void**) &dVals, totNnz * sizeof(DTYPE)), 0);
	checkCuda(hipMalloc((void**) &dInds0, totNnz * sizeof(ITYPE)), 0);
	checkCuda(hipMalloc((void**) &dInds1, totNnz * sizeof(ITYPE)), 0);
	checkCuda(hipMalloc((void**) &dInds2, totNnz * sizeof(ITYPE)), 0);

	for (int tile = 0; tile < Opt.nTile; ++tile){
		
		if(tile > 0) 
			dLoc += TiledX[tile-1].totNnz;

		checkCuda(hipMemcpy(dVals + dLoc, &(TiledX[tile].vals[0]), TiledX[tile].totNnz * sizeof(DTYPE),hipMemcpyHostToDevice), 0);
		checkCuda(hipMemcpy(dInds0 + dLoc, &(TiledX[tile].inds[mode0][0]), TiledX[tile].totNnz * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
		checkCuda(hipMemcpy(dInds1 + dLoc, &(TiledX[tile].inds[mode1][0]), TiledX[tile].totNnz * sizeof(ITYPE) ,hipMemcpyHostToDevice), 0);
		checkCuda(hipMemcpy(dInds2 + dLoc, &(TiledX[tile].inds[mode2][0]), TiledX[tile].totNnz * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
	}

	// //Matrices
	DTYPE *dU0, *dU1, *dU2;	
	checkCuda(hipMalloc((void**) &dU0, U[mode0].nRows * U[mode0].nCols * sizeof(DTYPE)), 0);
	checkCuda(hipMalloc((void**) &dU1, U[mode1].nRows * U[mode1].nCols * sizeof(DTYPE)), 0);
	checkCuda(hipMalloc((void**) &dU2, U[mode2].nRows * U[mode2].nCols * sizeof(DTYPE)), 0);

	hipMemset(dU0, 0,  U[mode0].nRows * U[mode0].nCols * sizeof(DTYPE));
	checkCuda(hipMemcpy(dU1, &(U[mode1].vals[0]), U[mode1].nRows * U[mode1].nCols * sizeof(DTYPE), hipMemcpyHostToDevice), 0);
	checkCuda(hipMemcpy(dU2, &(U[mode2].vals[0]), U[mode2].nRows * U[mode2].nCols * sizeof(DTYPE), hipMemcpyHostToDevice), 0);
	
	// BLOCK and GRID
	int BLOCKSIZE = 128;
	dim3 block(BLOCKSIZE, 1, 1), grid(1, 1, 1);
	
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float mili = 0, GPUTime = 0;

	// CUDA call
	dLoc = 0;
	for (int tile = 0; tile < Opt.nTile; ++tile){
		
		if(tile > 0) 
			dLoc += TiledX[tile-1].totNnz;

		cout << "Tile " << tile << " launched.. "<<endl;
		
		grid.x = (32 * TiledX[tile].totNnz + BLOCKSIZE - 1) / BLOCKSIZE;

		checkCuda(hipEventRecord(start), __LINE__);
		mttkrp_COO_kernel<<<grid, block>>>(dVals + dLoc, dInds0 + dLoc, dInds1 + dLoc, dInds2 + dLoc, TiledX[tile].totNnz, dU0, dU1, dU2,
								mode, R); 
	
		checkCuda(hipEventRecord(stop), __LINE__);
	    hipEventSynchronize(stop);
	    checkCuda(hipEventElapsedTime(&mili, start, stop), __LINE__);
	    hipDeviceSynchronize();
	    cout << "Tile: " << tile << " - time " << mili << "ms"<< endl;
	    GPUTime += mili;
	   
	}
	cout << "COO GPU - time " << GPUTime << "ms"<< endl;

	// check correctness
	checkCuda(hipMemcpy(&U[mode0].vals[0], dU0, U[mode0].nRows * U[mode0].nCols * sizeof(DTYPE), hipMemcpyDeviceToHost), 0);
	hipFree(dVals); 
	hipFree(dU0); hipFree(dU1); hipFree(dU2);
	hipFree(dInds0); hipFree(dInds1); hipFree(dInds2);

	return 0;
}

int MTTKRP_B_HCSR_GPU(TiledTensor *TiledX, Matrix *U, const Options &Opt){
	
	/*choosing kernel type:
	false: B-CSF- IPDPS work, true: parallelism at fiber level, call slc_atomic_fbrlblpar function*/
	bool slcAtomicFbrLvlPar =  false;

	/* Allocate and memcpy GPU memory */
	//Tensor
	ITYPE *dInds2, *dInds3, *dfbrPtr0, *dfbrIdx0, *dfbrPtr1, *dfbrIdx1, *dFbrPtr2, *dFbrIdx2, *dSlcMapperBin, *dFbrLikeSlcInds;
	DTYPE *dVals;
	ITYPE dLoc = 0, dSlcLoc = 0, dSlcIdxLoc = 0, dFbrLoc =0,  dFbrIdxLoc =0, dBinLoc = 0, dFbrLoc2 =0;
	ITYPE totNnz = 0, totSlcPtr = 0, totSlcIdx = 0, totFbrPtr = 0, totFbrIdx = 0, totFbrPtr2 = 0;

	// // All tile same mode
	ITYPE mode0 = TiledX[0].modeOrder[0];
    ITYPE mode1 = TiledX[0].modeOrder[1];
    ITYPE mode2 = TiledX[0].modeOrder[2];
    ITYPE mode3 =((TiledX[0].ndims == 4) ? TiledX[0].modeOrder[3] : 0) ;

	for (int tile = 0; tile < Opt.nTile; ++tile){
		totNnz += TiledX[tile].totNnz;
		totSlcPtr += TiledX[tile].fbrPtr[0].size() ;
		totSlcIdx += TiledX[tile].fbrIdx[0].size() ;
		totFbrPtr += TiledX[tile].fbrPtr[1].size() ;
		totFbrIdx += TiledX[tile].fbrIdx[1].size() ;
		totFbrPtr2 += ((TiledX[tile].ndims == 4) ? TiledX[tile].fbrPtr[2].size() : 0) ;
	}

	double t0 = seconds();
	checkCuda(hipMalloc((void**) &dVals, totNnz * sizeof(DTYPE)), 0);
	checkCuda(hipMalloc((void**) &dfbrPtr0, totSlcPtr * sizeof(ITYPE)), 0);
	checkCuda(hipMalloc((void**) &dfbrIdx0, totSlcIdx * sizeof(ITYPE)), 0);
	checkCuda(hipMalloc((void**) &dSlcMapperBin, totSlcPtr * sizeof(ITYPE)), 0);
	checkCuda(hipMalloc((void**) &dfbrPtr1, totFbrPtr * sizeof(ITYPE)), 0);
	checkCuda(hipMalloc((void**) &dfbrIdx1, totFbrIdx * sizeof(ITYPE)), 0);
	checkCuda(hipMalloc((void**) &dFbrLikeSlcInds, totFbrIdx * sizeof(ITYPE)), 0);

	if(TiledX[0].ndims == 3)
		checkCuda(hipMalloc((void**) &dInds2, totNnz * sizeof(ITYPE)), 0);

	if(TiledX[0].ndims == 4){
		checkCuda(hipMalloc((void**) &dFbrIdx2, totFbrPtr2 * sizeof(ITYPE)), 0);
		checkCuda(hipMalloc((void**) &dFbrPtr2, totFbrPtr2 * sizeof(ITYPE)), 0);
		checkCuda(hipMalloc((void**) &dInds3, totNnz * sizeof(ITYPE)), 0);
	}

	/* cuda memcopy for tiled parts*/
	for (int tile = 0; tile < Opt.nTile; ++tile){	
		if(tile > 0) {
			dLoc += TiledX[tile-1].totNnz;
			dSlcLoc += TiledX[tile - 1].fbrPtr[0].size(); // all tile same
			dSlcIdxLoc += TiledX[tile - 1].fbrIdx[0].size(); 
			dFbrLoc += TiledX[tile - 1].fbrPtr[1].size();
			dFbrIdxLoc += TiledX[tile - 1].fbrIdx[1].size();
			dFbrLoc2 += ((TiledX[tile].ndims == 4) ? TiledX[tile - 1].fbrPtr[2].size() : 0) ;
		}

		checkCuda(hipMemcpy(dVals + dLoc, &(TiledX[tile].vals[0]), TiledX[tile].totNnz * sizeof(DTYPE),hipMemcpyHostToDevice), 0);
		checkCuda(hipMemcpy(dfbrPtr0 + dSlcLoc, &(TiledX[tile].fbrPtr[0][0]), TiledX[tile].fbrPtr[0].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
		checkCuda(hipMemcpy(dfbrIdx0 + dSlcIdxLoc, &(TiledX[tile].fbrIdx[0][0]), TiledX[tile].fbrIdx[0].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
		checkCuda(hipMemcpy(dfbrPtr1 + dFbrLoc, &(TiledX[tile].fbrPtr[1][0]), TiledX[tile].fbrPtr[1].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
		checkCuda(hipMemcpy(dfbrIdx1 + dFbrIdxLoc, &(TiledX[tile].fbrIdx[1][0]), TiledX[tile].fbrIdx[1].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
		
		if(slcAtomicFbrLvlPar)
			checkCuda(hipMemcpy(dFbrLikeSlcInds + dFbrIdxLoc, &(TiledX[tile].fbrLikeSlcInds[0]), TiledX[tile].fbrIdx[1].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
	
		if(TiledX[tile].ndims == 3)
			checkCuda(hipMemcpy(dInds2 + dLoc, &(TiledX[tile].inds[TiledX[tile].modeOrder[2]][0]), TiledX[tile].totNnz * sizeof(ITYPE),hipMemcpyHostToDevice), 0);			

		if(TiledX[tile].ndims == 4){			
			checkCuda(hipMemcpy(dFbrPtr2 + dFbrLoc2, &(TiledX[tile].fbrPtr[2][0]), TiledX[tile].fbrPtr[2].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
			checkCuda(hipMemcpy(dFbrIdx2 + dFbrLoc2, &(TiledX[tile].fbrIdx[2][0]), TiledX[tile].fbrIdx[2].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
			checkCuda(hipMemcpy(dInds3 + dLoc, &(TiledX[tile].inds[mode3][0]), TiledX[tile].totNnz * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
		}

		dBinLoc = 0;
		for (int bin = 0; bin < Opt.nBin; ++bin){

			if(bin > 0)
				dBinLoc += TiledX[tile].slcMapperBin[bin-1].size();

		    checkCuda(hipMemcpy(dSlcMapperBin + dSlcIdxLoc + dBinLoc, &(TiledX[tile].slcMapperBin[bin][0]), TiledX[tile].slcMapperBin[bin].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
		}
	}
	float tnsMemcpyTime = seconds() - t0;

	t0 = seconds();
	// //Matrices
	DTYPE *dU0, *dU1, *dU2, *dU3;	
	checkCuda(hipMalloc((void**) &dU0, U[mode0].nRows * U[mode0].nCols * sizeof(DTYPE)), 0);
	checkCuda(hipMalloc((void**) &dU1, U[mode1].nRows * U[mode1].nCols * sizeof(DTYPE)), 0);
	checkCuda(hipMalloc((void**) &dU2, U[mode2].nRows * U[mode2].nCols * sizeof(DTYPE)), 0);

	hipMemset(dU0, 0,  U[mode0].nRows * U[mode0].nCols * sizeof(DTYPE));
	checkCuda(hipMemcpy(dU1, &(U[mode1].vals[0]), U[mode1].nRows * U[mode1].nCols * sizeof(DTYPE), hipMemcpyHostToDevice), 0);
	checkCuda(hipMemcpy(dU2, &(U[mode2].vals[0]), U[mode2].nRows * U[mode2].nCols * sizeof(DTYPE), hipMemcpyHostToDevice), 0);

	float mtxMemcpyTime = seconds() - t0;

	// cout << "tns and mtx memcopy time: " << tnsMemcpyTime <<", " << mtxMemcpyTime<< endl;
	
	if(TiledX[0].ndims == 4){
		checkCuda(hipMalloc((void**) &dU3, U[mode3].nRows * U[mode3].nCols * sizeof(DTYPE)), 0);
		checkCuda(hipMemcpy(dU3, &(U[mode3].vals[0]), U[mode3].nRows * U[mode3].nCols * sizeof(DTYPE), hipMemcpyHostToDevice), 0);
	}

	// BLOCK and GRID
	int BLOCKSIZE = 512;
	unsigned int rowInATB = BLOCKSIZE / (Opt.warpPerSlice*32); 

	if(Opt.warpPerSlice * 32 > BLOCKSIZE){
		cout << "BLOCKSIZE is smaller than work per slice! Increase BLOCKSIZE." << endl;
		exit(0);
	}

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipStream_t streams[Opt.nBin];
    float mili = 0, GPUTime = 0, CPUtimer = 0, allModeGPUTime = 0;

    int smallBinEndsAt = 5;

    /* Warp per slice and threadblock per size */
    int *warpPerSlc = new int[Opt.nBin];
    int *logOfWarpPerSlc = new int[Opt.nBin];
    int *TbPerSlc = new int[Opt.nBin];
    int *logOfTbPerSlc = new int[Opt.nBin];

    for (int bin = 0; bin < Opt.nBin ; ++bin){
    	
    	TbPerSlc[bin] = 1;
		warpPerSlc[bin] = ((bin > 0) ? 2 << (bin - 1) : 1);
		
		if(warpPerSlc[bin] > 16)		
			warpPerSlc[bin] = 16;

		logOfWarpPerSlc[bin] = log2(warpPerSlc[bin]);

		TbPerSlc[bin] = 1;
		logOfTbPerSlc[bin] = 0;
		
		if (bin >= smallBinEndsAt){
		
			TbPerSlc[bin] = 1 << (bin - smallBinEndsAt + 1); // 1st big bin starts with 1 TB 1 << 1 not 1 << 5
			if(TbPerSlc[bin] > 32) TbPerSlc[bin] = 32;		
			logOfTbPerSlc[bin] = log2(TbPerSlc[bin]);

			warpPerSlc[bin] = 16;
			logOfWarpPerSlc[bin] = 4;
		}
    }

    // TBD: change warpPerSlc to warpPerSlc[bin] and all
	int slcPerTb = 1;

	dLoc = 0, dSlcLoc = 0, dSlcIdxLoc = 0; dFbrLoc =0, dFbrIdxLoc = 0, dFbrLoc2= 0;

	for (int bin = 0; bin < Opt.nBin; ++bin)
		hipStreamCreate(&streams[bin]);

	/*MTTKRP on Opt.mode*/
	int MTTKRPmode = mode0;//Opt.mode;

	for (int tile = 0; tile < Opt.nTile; ++tile){

		dBinLoc = 0;
		
		if(tile > 0) {
			dLoc += TiledX[tile-1].totNnz;
			dSlcLoc += TiledX[tile - 1].fbrPtr[0].size(); 
			dSlcIdxLoc += TiledX[tile - 1].fbrIdx[0].size(); 
			dFbrLoc += TiledX[tile - 1].fbrPtr[1].size();
			dFbrIdxLoc += TiledX[tile - 1].fbrIdx[1].size();
			dFbrLoc2 += ((TiledX[0].ndims == 4) ? TiledX[tile - 1].fbrPtr[2].size() : 0) ;
		}

		BLOCKSIZE = 512;
		dim3 block(BLOCKSIZE, 1, 1), grid(1, 1, 1);

		int smallBinEndsAt = 5;
		int slcPerTb = 0;

		int warpPerFbr = 1;//BLOCKSIZE/32;//1;//Opt.warpPerSlice;//4;//;
		int logOfWarpPerFbr = log2(warpPerFbr);
		int bin = 0;
		int fbrPerWarp = BLOCKSIZE/32; // dont overflow TB
		int logOfFbrPerWarp = log2(fbrPerWarp );

		grid.x = ( warpPerFbr * 32 * ((TiledX[tile].nFibers+fbrPerWarp-1)/fbrPerWarp) + BLOCKSIZE - 1) / BLOCKSIZE;

		double t0 = seconds();
		cuda_timer_start(start);
		
		if(slcAtomicFbrLvlPar)
			mttkrp_MIHCSR_kernel_slc_atomic_fbrLvlPar<<<grid, block, 0, streams[bin]>>>(dVals + dLoc, dFbrLikeSlcInds + dFbrLoc, 
				dInds2 + dLoc, dfbrPtr0 + dSlcLoc, dfbrPtr1 + dFbrLoc,  dfbrIdx1 + dFbrLoc, TiledX[tile].nFibers, 
				dU0, dU1, dU2, Opt.mode, Opt.R, warpPerFbr, logOfWarpPerFbr, fbrPerWarp, logOfFbrPerWarp);
		
		else{

			for (int bin = 0; bin < Opt.nBin ; ++bin){

				if(bin < smallBinEndsAt){
					
					ITYPE shSize = 0;//slcPerTb * 32 * sizeof(DTYPE); slcPerTb = 16 / warpPerSlc[bin];

					dBinLoc += ((bin > 0) ? TiledX[tile].slcMapperBin[bin-1].size() : 0);

					grid.x = ( TbPerSlc[bin] * warpPerSlc[bin] * 32 * TiledX[tile].slcMapperBin[bin].size() + BLOCKSIZE - 1) / BLOCKSIZE;

					if(TiledX[0].ndims == 3)
						mttkrp_HCSR_kernel_smllBin<<<grid, block, shSize , streams[bin]>>>(dVals + dLoc, dfbrIdx0 + dSlcIdxLoc, dSlcMapperBin + dSlcIdxLoc + dBinLoc, 
						dInds2 + dLoc, dfbrPtr0 + dSlcLoc, dfbrPtr1 + dFbrLoc,  dfbrIdx1 + dFbrLoc, TiledX[tile].slcMapperBin[bin].size(), 
						dU0, dU1, dU2, Opt.mode, Opt.R, warpPerSlc[bin], logOfWarpPerSlc[bin], TbPerSlc[bin], logOfTbPerSlc[bin]); 
					else
						mttkrp_HCSR_kernel_smllBin_4D<<<grid, block, shSize , streams[bin]>>>(dVals + dLoc, dfbrIdx0 + dSlcIdxLoc, dSlcMapperBin + dSlcIdxLoc + dBinLoc, 
						dInds3 + dLoc, dfbrPtr0 + dSlcLoc, dfbrPtr1 + dFbrLoc, dfbrIdx1 + dFbrIdxLoc, dFbrPtr2 + dFbrLoc2, dFbrIdx2 + dFbrLoc2, TiledX[tile].slcMapperBin[bin].size(), 
						dU0, dU1, dU2, dU3, Opt.mode, Opt.R, warpPerSlc[bin], logOfWarpPerSlc[bin], TbPerSlc[bin], logOfTbPerSlc[bin]); 
				}
				
				// Processing heavy bin.. multiple TB per slice
				else{

					dBinLoc += TiledX[tile].slcMapperBin[bin-1].size();
							
					grid.x = (TbPerSlc[bin] * warpPerSlc[bin] * 32 * TiledX[tile].slcMapperBin[bin].size() + BLOCKSIZE - 1) / BLOCKSIZE;
					
						if(TiledX[0].ndims == 3)
							mttkrp_HCSR_kernel_hvyBin<<<grid, block, 0, streams[bin]>>>(dVals + dLoc, dfbrIdx0 + dSlcIdxLoc, dSlcMapperBin + dSlcIdxLoc + dBinLoc, 
							dInds2 + dLoc, dfbrPtr0 + dSlcLoc, dfbrPtr1 + dFbrLoc,  dfbrIdx1 + dFbrLoc, TiledX[tile].slcMapperBin[bin].size(), 
							dU0, dU1, dU2, Opt.mode, Opt.R, warpPerSlc[bin], logOfWarpPerSlc[bin],  TbPerSlc[bin], logOfTbPerSlc[bin]); 
						else
							mttkrp_HCSR_kernel_hvyBin_4D<<<grid, block, 0, streams[bin]>>>(dVals + dLoc, dfbrIdx0 + dSlcIdxLoc, dSlcMapperBin + dSlcIdxLoc + dBinLoc, 
							dInds3 + dLoc, dfbrPtr0 + dSlcLoc, dfbrPtr1 + dFbrLoc, dfbrIdx1 + dFbrIdxLoc, dFbrPtr2 + dFbrLoc2, dFbrIdx2 + dFbrLoc2, TiledX[tile].slcMapperBin[bin].size(), 
							dU0, dU1, dU2, dU3, Opt.mode, Opt.R, warpPerSlc[bin], logOfWarpPerSlc[bin],  TbPerSlc[bin], logOfTbPerSlc[bin]); 
				}
			}
		}

		cuda_timer_stop(start, stop, mili);
	    CPUtimer += seconds() - t0;
	    GPUTime += mili;

	    if(Opt.verbose){
	    	cout << "Tile: " << tile << " - time: " << mili << "ms";
	    	cout <<" nnz: " << TiledX[tile].totNnz << " nFibers: "
	    	<< TiledX[tile].fbrPtr[1].size() << " nSlc " << TiledX[tile].fbrIdx[0].size() << " ";
			cout << endl;
		} 
		
	}
	allModeGPUTime += GPUTime;
	cout << "B-CSF-GPU-mode " << MTTKRPmode <<" :" << GPUTime << "," << endl;

	for (int bin = 0; bin < Opt.nBin; ++bin)
		hipStreamDestroy(streams[bin]);

	// check correctness
	checkCuda(hipMemcpy(&U[mode0].vals[0], dU0, U[mode0].nRows * U[mode0].nCols * sizeof(DTYPE), hipMemcpyDeviceToHost), 0);

	hipFree(dVals); 
	hipFree(dU0); hipFree(dU1); hipFree(dU2); hipFree(dU3);
	hipFree(dfbrIdx0); hipFree(dInds2); hipFree(dInds3); 
	hipFree(dfbrIdx0); hipFree(dfbrIdx1); hipFree(dFbrIdx2);
	hipFree(dfbrPtr0); hipFree(dfbrPtr1); hipFree(dFbrPtr2);
	hipFree(dFbrLikeSlcInds);

	return 0;
}

int MTTKRP_ONE_HCSR_GPU(TiledTensor *TiledX, Matrix *U, const Options &Opt){
	
	bool performMTTKRPMode = true, performMTTKRPnMode = true, performMTTKRPnnMode = true;
	
	/* Allocate and memcpy GPU memory */
	//Tensor
	ITYPE *dInds2, *dInds3, *dfbrPtr0, *dfbrIdx0, *dfbrPtr1, *dfbrIdx1, *dFbrPtr2, *dFbrIdx2, *dSlcMapperBin, *dFbrLikeSlcInds;
	DTYPE *dVals;
	ITYPE dLoc = 0, dSlcLoc = 0, dSlcIdxLoc = 0, dFbrLoc =0,  dFbrIdxLoc =0, dBinLoc = 0, dFbrLoc2 =0;
	ITYPE totNnz = 0, totSlcPtr = 0, totSlcIdx = 0, totFbrPtr = 0, totFbrIdx = 0, totFbrPtr2 = 0;

	// // All tile same mode
	ITYPE mode0 = TiledX[0].modeOrder[0];
    ITYPE mode1 = TiledX[0].modeOrder[1];
    ITYPE mode2 = TiledX[0].modeOrder[2];
    ITYPE mode3 =((TiledX[0].ndims == 4) ? TiledX[0].modeOrder[3] : 0) ;

	for (int tile = 0; tile < Opt.nTile; ++tile){
		totNnz += TiledX[tile].totNnz;
		totSlcPtr += TiledX[tile].fbrPtr[0].size() ;
		totSlcIdx += TiledX[tile].fbrIdx[0].size() ;
		totFbrPtr += TiledX[tile].fbrPtr[1].size() ;
		totFbrIdx += TiledX[tile].fbrIdx[1].size() ;
		totFbrPtr2 += ((TiledX[tile].ndims == 4) ? TiledX[tile].fbrPtr[2].size() : 0) ;
	}

	double t0 = seconds();
	checkCuda(hipMalloc((void**) &dVals, totNnz * sizeof(DTYPE)), 0);
	checkCuda(hipMalloc((void**) &dfbrPtr0, totSlcPtr * sizeof(ITYPE)), 0);
	checkCuda(hipMalloc((void**) &dfbrIdx0, totSlcIdx * sizeof(ITYPE)), 0);
	checkCuda(hipMalloc((void**) &dSlcMapperBin, totSlcPtr * sizeof(ITYPE)), 0);
	checkCuda(hipMalloc((void**) &dfbrPtr1, totFbrPtr * sizeof(ITYPE)), 0);
	checkCuda(hipMalloc((void**) &dfbrIdx1, totFbrIdx * sizeof(ITYPE)), 0);
	checkCuda(hipMalloc((void**) &dFbrLikeSlcInds, totFbrIdx * sizeof(ITYPE)), 0);

	if(TiledX[0].ndims == 3)
		checkCuda(hipMalloc((void**) &dInds2, totNnz * sizeof(ITYPE)), 0);

	if(TiledX[0].ndims == 4){
		checkCuda(hipMalloc((void**) &dFbrIdx2, totFbrPtr2 * sizeof(ITYPE)), 0);
		checkCuda(hipMalloc((void**) &dFbrPtr2, totFbrPtr2 * sizeof(ITYPE)), 0);
		checkCuda(hipMalloc((void**) &dInds3, totNnz * sizeof(ITYPE)), 0);
	}

	/* cuda memcopy for tiled parts*/
	for (int tile = 0; tile < Opt.nTile; ++tile){	
		if(tile > 0) {
			dLoc += TiledX[tile-1].totNnz;
			dSlcLoc += TiledX[tile - 1].fbrPtr[0].size(); // all tile same
			dSlcIdxLoc += TiledX[tile - 1].fbrIdx[0].size(); 
			dFbrLoc += TiledX[tile - 1].fbrPtr[1].size();
			dFbrIdxLoc += TiledX[tile - 1].fbrIdx[1].size();
			dFbrLoc2 += ((TiledX[tile].ndims == 4) ? TiledX[tile - 1].fbrPtr[2].size() : 0) ;
		}

		checkCuda(hipMemcpy(dVals + dLoc, &(TiledX[tile].vals[0]), TiledX[tile].totNnz * sizeof(DTYPE),hipMemcpyHostToDevice), 0);
		checkCuda(hipMemcpy(dfbrPtr0 + dSlcLoc, &(TiledX[tile].fbrPtr[0][0]), TiledX[tile].fbrPtr[0].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
		checkCuda(hipMemcpy(dfbrIdx0 + dSlcIdxLoc, &(TiledX[tile].fbrIdx[0][0]), TiledX[tile].fbrIdx[0].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
		checkCuda(hipMemcpy(dfbrPtr1 + dFbrLoc, &(TiledX[tile].fbrPtr[1][0]), TiledX[tile].fbrPtr[1].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
		checkCuda(hipMemcpy(dfbrIdx1 + dFbrIdxLoc, &(TiledX[tile].fbrIdx[1][0]), TiledX[tile].fbrIdx[1].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
		checkCuda(hipMemcpy(dFbrLikeSlcInds + dFbrIdxLoc, &(TiledX[tile].fbrLikeSlcInds[0]), TiledX[tile].fbrIdx[1].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
	
		if(TiledX[tile].ndims == 3)
			checkCuda(hipMemcpy(dInds2 + dLoc, &(TiledX[tile].inds[TiledX[tile].modeOrder[2]][0]), TiledX[tile].totNnz * sizeof(ITYPE),hipMemcpyHostToDevice), 0);			

		if(TiledX[tile].ndims == 4){			
			checkCuda(hipMemcpy(dFbrPtr2 + dFbrLoc2, &(TiledX[tile].fbrPtr[2][0]), TiledX[tile].fbrPtr[2].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
			checkCuda(hipMemcpy(dFbrIdx2 + dFbrLoc2, &(TiledX[tile].fbrIdx[2][0]), TiledX[tile].fbrIdx[2].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
			checkCuda(hipMemcpy(dInds3 + dLoc, &(TiledX[tile].inds[mode3][0]), TiledX[tile].totNnz * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
		}

		dBinLoc = 0;
		for (int bin = 0; bin < Opt.nBin; ++bin){

			if(bin > 0)
				dBinLoc += TiledX[tile].slcMapperBin[bin-1].size();

		    checkCuda(hipMemcpy(dSlcMapperBin + dSlcIdxLoc + dBinLoc, &(TiledX[tile].slcMapperBin[bin][0]), TiledX[tile].slcMapperBin[bin].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
		}
	}
	float tnsMemcpyTime = seconds() - t0;

	t0 = seconds();
	// //Matrices
	DTYPE *dU0, *dU1, *dU2, *dU3;	
	checkCuda(hipMalloc((void**) &dU0, U[mode0].nRows * U[mode0].nCols * sizeof(DTYPE)), 0);
	checkCuda(hipMalloc((void**) &dU1, U[mode1].nRows * U[mode1].nCols * sizeof(DTYPE)), 0);
	checkCuda(hipMalloc((void**) &dU2, U[mode2].nRows * U[mode2].nCols * sizeof(DTYPE)), 0);

	hipMemset(dU0, 0,  U[mode0].nRows * U[mode0].nCols * sizeof(DTYPE));
	checkCuda(hipMemcpy(dU1, &(U[mode1].vals[0]), U[mode1].nRows * U[mode1].nCols * sizeof(DTYPE), hipMemcpyHostToDevice), 0);
	checkCuda(hipMemcpy(dU2, &(U[mode2].vals[0]), U[mode2].nRows * U[mode2].nCols * sizeof(DTYPE), hipMemcpyHostToDevice), 0);

	float mtxMemcpyTime = seconds() - t0;

	// cout << "tns and mtx memcopy time: " << tnsMemcpyTime <<", " << mtxMemcpyTime<< endl;
	
	if(TiledX[0].ndims == 4){
		checkCuda(hipMalloc((void**) &dU3, U[mode3].nRows * U[mode3].nCols * sizeof(DTYPE)), 0);
		checkCuda(hipMemcpy(dU3, &(U[mode3].vals[0]), U[mode3].nRows * U[mode3].nCols * sizeof(DTYPE), hipMemcpyHostToDevice), 0);
	}

	// BLOCK and GRID
	int BLOCKSIZE = 512;
	unsigned int rowInATB = BLOCKSIZE / (Opt.warpPerSlice*32); 

	if(Opt.warpPerSlice * 32 > BLOCKSIZE){
		cout << "BLOCKSIZE is smaller than work per slice! Increase BLOCKSIZE." << endl;
		exit(0);
	}

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipStream_t streams[Opt.nBin];
    float mili = 0, GPUTime = 0, CPUtimer = 0, allModeGPUTime = 0;

    int smallBinEndsAt = 5;

    /* Warp per slice and threadblock per size */
    int *warpPerSlc = new int[Opt.nBin];
    int *logOfWarpPerSlc = new int[Opt.nBin];
    int *TbPerSlc = new int[Opt.nBin];
    int *logOfTbPerSlc = new int[Opt.nBin];

    for (int bin = 0; bin < Opt.nBin ; ++bin){
    	
    	TbPerSlc[bin] = 1;
		warpPerSlc[bin] = ((bin > 0) ? 2 << (bin - 1) : 1);
		
		if(warpPerSlc[bin] > 16)		
			warpPerSlc[bin] = 16;

		logOfWarpPerSlc[bin] = log2(warpPerSlc[bin]);

		TbPerSlc[bin] = 1;
		logOfTbPerSlc[bin] = 0;
		
		if (bin >= smallBinEndsAt){
		
			TbPerSlc[bin] = 1 << (bin - smallBinEndsAt + 1); // 1st big bin starts with 1 TB 1 << 1 not 1 << 5
			if(TbPerSlc[bin] > 32) TbPerSlc[bin] = 32;		
			logOfTbPerSlc[bin] = log2(TbPerSlc[bin]);

			warpPerSlc[bin] = 16;
			logOfWarpPerSlc[bin] = 4;
		}
    }

    // TBD: change warpPerSlc to warpPerSlc[bin] and all
	int slcPerTb = 1;

	dLoc = 0, dSlcLoc = 0, dSlcIdxLoc = 0; dFbrLoc =0, dFbrIdxLoc = 0, dFbrLoc2= 0;

	for (int bin = 0; bin < Opt.nBin; ++bin)
		hipStreamCreate(&streams[bin]);

	/*MTTKRP on Opt.mode*/
	int MTTKRPmode = mode0;//Opt.mode;

	if(performMTTKRPMode){
		
		for (int tile = 0; tile < Opt.nTile; ++tile){

			dBinLoc = 0;
			
			if(tile > 0) {
				dLoc += TiledX[tile-1].totNnz;
				dSlcLoc += TiledX[tile - 1].fbrPtr[0].size(); 
				dSlcIdxLoc += TiledX[tile - 1].fbrIdx[0].size(); 
				dFbrLoc += TiledX[tile - 1].fbrPtr[1].size();
				dFbrIdxLoc += TiledX[tile - 1].fbrIdx[1].size();
				dFbrLoc2 += ((TiledX[0].ndims == 4) ? TiledX[tile - 1].fbrPtr[2].size() : 0) ;
			}

			// BLOCKSIZE = 512;
			// dim3 block(BLOCKSIZE, 1, 1), grid(1, 1, 1);
			
			BLOCKSIZE = 512;
			dim3 block(BLOCKSIZE, 1, 1), grid(1, 1, 1);
			int smallBinEndsAt = 5;
			int slcPerTb = 0;
			int warpPerFbr = 1;//BLOCKSIZE/32;//1;//Opt.warpPerSlice;//4;//;
			int logOfWarpPerFbr = log2(warpPerFbr);
			int bin = 0;
			bool useLoop = false;
			int fbrPerWarp = 16; // dont overflow TB
			int logOfFbrPerWarp = log2(fbrPerWarp );

			// /* Like PARTI loop */ = 
			if(useLoop)
				grid.x = Opt.gridSize;// 32768*16; 
			else 
				grid.x = ( warpPerFbr * 32 * ((TiledX[tile].nFibers+fbrPerWarp-1)/fbrPerWarp) + BLOCKSIZE - 1) / BLOCKSIZE;

			double t0 = seconds();
			cuda_timer_start(start);

			mttkrp_MIHCSR_kernel_slc_atomic_fbrLvlPar<<<grid, block, 0, streams[bin]>>>(dVals + dLoc, dFbrLikeSlcInds + dFbrLoc, 
				dInds2 + dLoc, dfbrPtr0 + dSlcLoc, dfbrPtr1 + dFbrLoc,  dfbrIdx1 + dFbrLoc, TiledX[tile].nFibers, 
				dU0, dU1, dU2, Opt.mode, Opt.R, warpPerFbr, logOfWarpPerFbr, fbrPerWarp, logOfFbrPerWarp);
			
			// for (int bin = 0; bin < Opt.nBin ; ++bin){

			// 	if(bin < smallBinEndsAt){
					
			// 		ITYPE shSize = 0;//slcPerTb * 32 * sizeof(DTYPE); slcPerTb = 16 / warpPerSlc[bin];

			// 		dBinLoc += ((bin > 0) ? TiledX[tile].slcMapperBin[bin-1].size() : 0);

			// 		grid.x = ( TbPerSlc[bin] * warpPerSlc[bin] * 32 * TiledX[tile].slcMapperBin[bin].size() + BLOCKSIZE - 1) / BLOCKSIZE;

			// 		if(TiledX[0].ndims == 3)
			// 			mttkrp_HCSR_kernel_smllBin<<<grid, block, shSize , streams[bin]>>>(dVals + dLoc, dfbrIdx0 + dSlcIdxLoc, dSlcMapperBin + dSlcIdxLoc + dBinLoc, 
			// 			dInds2 + dLoc, dfbrPtr0 + dSlcLoc, dfbrPtr1 + dFbrLoc,  dfbrIdx1 + dFbrLoc, TiledX[tile].slcMapperBin[bin].size(), 
			// 			dU0, dU1, dU2, Opt.mode, Opt.R, warpPerSlc[bin], logOfWarpPerSlc[bin], TbPerSlc[bin], logOfTbPerSlc[bin]); 
			// 		else
			// 			mttkrp_HCSR_kernel_smllBin_4D<<<grid, block, shSize , streams[bin]>>>(dVals + dLoc, dfbrIdx0 + dSlcIdxLoc, dSlcMapperBin + dSlcIdxLoc + dBinLoc, 
			// 			dInds3 + dLoc, dfbrPtr0 + dSlcLoc, dfbrPtr1 + dFbrLoc, dfbrIdx1 + dFbrIdxLoc, dFbrPtr2 + dFbrLoc2, dFbrIdx2 + dFbrLoc2, TiledX[tile].slcMapperBin[bin].size(), 
			// 			dU0, dU1, dU2, dU3, Opt.mode, Opt.R, warpPerSlc[bin], logOfWarpPerSlc[bin], TbPerSlc[bin], logOfTbPerSlc[bin]); 
			// 	}
				
			// 	// Processing heavy bin.. multiple TB per slice
			// 	else{

			// 		dBinLoc += TiledX[tile].slcMapperBin[bin-1].size();
							
			// 		grid.x = (TbPerSlc[bin] * warpPerSlc[bin] * 32 * TiledX[tile].slcMapperBin[bin].size() + BLOCKSIZE - 1) / BLOCKSIZE;
					
			// 			if(TiledX[0].ndims == 3)
			// 				mttkrp_HCSR_kernel_hvyBin<<<grid, block, 0, streams[bin]>>>(dVals + dLoc, dfbrIdx0 + dSlcIdxLoc, dSlcMapperBin + dSlcIdxLoc + dBinLoc, 
			// 				dInds2 + dLoc, dfbrPtr0 + dSlcLoc, dfbrPtr1 + dFbrLoc,  dfbrIdx1 + dFbrLoc, TiledX[tile].slcMapperBin[bin].size(), 
			// 				dU0, dU1, dU2, Opt.mode, Opt.R, warpPerSlc[bin], logOfWarpPerSlc[bin],  TbPerSlc[bin], logOfTbPerSlc[bin]); 
			// 			else
			// 				mttkrp_HCSR_kernel_hvyBin_4D<<<grid, block, 0, streams[bin]>>>(dVals + dLoc, dfbrIdx0 + dSlcIdxLoc, dSlcMapperBin + dSlcIdxLoc + dBinLoc, 
			// 				dInds3 + dLoc, dfbrPtr0 + dSlcLoc, dfbrPtr1 + dFbrLoc, dfbrIdx1 + dFbrIdxLoc, dFbrPtr2 + dFbrLoc2, dFbrIdx2 + dFbrLoc2, TiledX[tile].slcMapperBin[bin].size(), 
			// 				dU0, dU1, dU2, dU3, Opt.mode, Opt.R, warpPerSlc[bin], logOfWarpPerSlc[bin],  TbPerSlc[bin], logOfTbPerSlc[bin]); 
			// 	}
			// }
			cuda_timer_stop(start, stop, mili);
		    CPUtimer += seconds() - t0;
		    GPUTime += mili;

		    if(Opt.verbose){
		    	cout << "Tile: " << tile << " - time: " << mili << "ms";
		    	cout <<" nnz: " << TiledX[tile].totNnz << " nFibers: "
		    	<< TiledX[tile].fbrPtr[1].size() << " nSlc " << TiledX[tile].fbrIdx[0].size() << " ";
				cout << endl;
			} 
		}
	}
	allModeGPUTime += GPUTime;
	cout << "singleCSF-GPU-mode " << MTTKRPmode <<" :" << GPUTime << "," << endl;

	/*next mode*/

	MTTKRPmode = mode1;//(1 + Opt.mode) % TiledX[0].ndims;

	if(performMTTKRPnMode){

		mili = 0, GPUTime = 0, CPUtimer = 0;
		dLoc = 0, dSlcLoc = 0, dSlcIdxLoc = 0; dFbrLoc =0, dFbrIdxLoc = 0, dFbrLoc2= 0;

		//  U[mode].vals is all 0.To pass correctness for now initializing to 2 again.
	    for(long r = 0; r < U[mode0].nRows; ++r){
	        for(long c = 0; c < U[mode0].nCols; ++c) // or u[mode].nCols 
	            U[mode0].vals[r * U[mode0].nCols + c] = mode0 + .5;//2 * drand48(); //1 ;//(r * R + c + 1); //
	    }

	    checkCuda(hipMemcpy(dU0, &(U[mode0].vals[0]), U[mode0].nRows * U[mode0].nCols * sizeof(DTYPE), hipMemcpyHostToDevice), 0);	
		hipMemset(dU1, 0,  U[mode1].nRows * U[mode1].nCols * sizeof(DTYPE));

		for (int tile = 0; tile < Opt.nTile; ++tile){

			dBinLoc = 0;
			
			if(tile > 0) {
				dLoc += TiledX[tile-1].totNnz;
				dSlcLoc += TiledX[tile - 1].fbrPtr[0].size(); 
				dSlcIdxLoc += TiledX[tile - 1].fbrIdx[0].size(); 
				dFbrLoc += TiledX[tile - 1].fbrPtr[1].size();
				dFbrIdxLoc += TiledX[tile - 1].fbrIdx[1].size();
				dFbrLoc2 += ((TiledX[0].ndims == 4) ? TiledX[tile - 1].fbrPtr[2].size() : 0) ;
			}
			// cout <<"might wanna change binning style and Block size, logWPC, COO like parallelism, allow mode sort" << endl;

			BLOCKSIZE = 128;//Opt.TBsize;
			dim3 block(BLOCKSIZE, 1, 1), grid(1, 1, 1);

			int smallBinEndsAt = 5;
			int slcPerTb = 0;
			int warpPerFbr = 1;//BLOCKSIZE/32;//1;//Opt.warpPerSlice;//4;//;
			int logOfWarpPerFbr = log2(warpPerFbr);
			int bin = 0;
			bool useLoop = false;

			// /* Like PARTI loop */ = 
			if(useLoop)
				grid.x = Opt.gridSize;// 32768*16; 
			else 
				grid.x = ( warpPerFbr * 32 * TiledX[tile].nFibers + BLOCKSIZE - 1) / BLOCKSIZE;

			// Process small bins.. accepts 2 slice 1 TB

			double t0 = seconds();
			cuda_timer_start(start);
			
			if(useLoop)
				mttkrp_MIHCSR_kernel_fbr_atomic_fbrLvlPar_loop<<<grid, block, 0, streams[bin]>>>(dVals + dLoc, dFbrLikeSlcInds + dFbrLoc, 
			dInds2 + dLoc, dfbrPtr0 + dSlcLoc, dfbrPtr1 + dFbrLoc,  dfbrIdx1 + dFbrLoc, TiledX[tile].nFibers, 
			dU1, dU2, dU0, Opt.mode, Opt.R, warpPerFbr, logOfWarpPerFbr);
			
			else
				mttkrp_MIHCSR_kernel_fbr_atomic_fbrLvlPar<<<grid, block, 0, streams[bin]>>>(dVals + dLoc, dFbrLikeSlcInds + dFbrLoc, 
			dInds2 + dLoc, dfbrPtr0 + dSlcLoc, dfbrPtr1 + dFbrLoc,  dfbrIdx1 + dFbrLoc, TiledX[tile].nFibers, 
			dU1, dU2, dU0, Opt.mode, Opt.R, warpPerFbr, logOfWarpPerFbr);

			cuda_timer_stop(start, stop, mili);
		    CPUtimer += seconds() - t0;
		    GPUTime += mili;

		    if(Opt.verbose){
		    	cout << "Tile: " << tile << " - time: " << mili << "ms";
		    	cout <<" nnz: " << TiledX[tile].totNnz << " nFibers: "
		    	<< TiledX[tile].fbrPtr[1].size() << " nSlc " << TiledX[tile].fbrIdx[0].size() << " ";
				cout << endl;
			} 
		}
		allModeGPUTime += GPUTime;
		cout << "singleCSF-GPU-mode " << MTTKRPmode <<" :" << GPUTime << "," << endl;
	} 

	/*next-next mode*/

	MTTKRPmode = mode2;//(2 + Opt.mode) % TiledX[0].ndims;

	if(performMTTKRPnnMode){

		mili = 0, GPUTime = 0, CPUtimer = 0;
		dLoc = 0, dSlcLoc = 0, dSlcIdxLoc = 0; dFbrLoc =0, dFbrIdxLoc = 0, dFbrLoc2= 0;

		// MTTKRP on mode mode 1 changed DU1. To pass correctness for now initializing to 2 again.

	    for(long r = 0; r < U[mode1].nRows; ++r){
	        for(long c = 0; c < U[mode1].nCols; ++c) // or u[mode].nCols 
	            U[mode1].vals[r * U[mode1].nCols + c] = mode1 + .5;// drand48(); //1 ;//(r * R + c + 1); //
	    }
	    for(long r = 0; r < U[mode0].nRows; ++r){
	        for(long c = 0; c < U[mode0].nCols; ++c) // or u[mode].nCols 
	            U[mode0].vals[r * U[mode0].nCols + c] = mode0 + .5;//2 * drand48(); //1 ;//(r * R + c + 1); //
	    }
			checkCuda(hipMemcpy(dU0, &(U[mode0].vals[0]), U[mode0].nRows * U[mode0].nCols * sizeof(DTYPE), hipMemcpyHostToDevice), 0);	
	    checkCuda(hipMemcpy(dU1, &(U[mode1].vals[0]), U[mode1].nRows * U[mode1].nCols * sizeof(DTYPE), hipMemcpyHostToDevice), 0);	
		hipMemset(dU2, 0,  U[mode2].nRows * U[mode2].nCols * sizeof(DTYPE));

		for (int tile = 0; tile < Opt.nTile; ++tile){

			dBinLoc = 0;
			
			if(tile > 0) {
				dLoc += TiledX[tile-1].totNnz;
				dSlcLoc += TiledX[tile - 1].fbrPtr[0].size(); 
				dSlcIdxLoc += TiledX[tile - 1].fbrIdx[0].size(); 
				dFbrLoc += TiledX[tile - 1].fbrPtr[1].size();
				dFbrIdxLoc += TiledX[tile - 1].fbrIdx[1].size();
				dFbrLoc2 += ((TiledX[0].ndims == 4) ? TiledX[tile - 1].fbrPtr[2].size() : 0) ;
			}

			BLOCKSIZE = 128; ///Opt.TBsize;
			dim3 block(BLOCKSIZE, 1, 1), grid(1, 1, 1);

			bool useLoop = false;
			int smallBinEndsAt = 5;
			int slcPerTb = 0;
			int warpPerFbr = 1;//Opt.warpPerSlice;//4;//;BLOCKSIZE/32;//
			int logOfWarpPerFbr = log2(warpPerFbr);
			int bin = 0;
			

			// /* Like PARTI loop */ = 
			if(useLoop)
				grid.x = Opt.gridSize;// 32768;
			else 
				grid.x = ( warpPerFbr * 32 * TiledX[tile].nFibers + BLOCKSIZE - 1) / BLOCKSIZE;
			// cout <<"might wanna change binning style and Block size, allow mode sort" << endl;

			
			double t0 = seconds();
			cuda_timer_start(start);

			if(useLoop)
				mttkrp_MIHCSR_kernel_all_atomic_fbrLvlPar_loop<<<grid, block, 0, streams[bin]>>>(dVals + dLoc, dFbrLikeSlcInds + dFbrLoc, 
			dInds2 + dLoc, dfbrPtr0 + dSlcLoc, dfbrPtr1 + dFbrLoc,  dfbrIdx1 + dFbrLoc, TiledX[tile].nFibers, 
			dU2, dU0, dU1, Opt.mode, Opt.R, warpPerFbr, logOfWarpPerFbr);
			
			else
				mttkrp_MIHCSR_kernel_all_atomic_fbrLvlPar<<<grid, block, 0, streams[bin]>>>(dVals + dLoc, dFbrLikeSlcInds + dFbrLoc, 
			dInds2 + dLoc, dfbrPtr0 + dSlcLoc, dfbrPtr1 + dFbrLoc,  dfbrIdx1 + dFbrLoc, TiledX[tile].nFibers, 
			dU2, dU0, dU1, Opt.mode, Opt.R, warpPerFbr, logOfWarpPerFbr); 
			
			cuda_timer_stop(start, stop, mili);
		    CPUtimer += seconds() - t0;
		    GPUTime += mili;

		    if(Opt.verbose){
		    	cout << "Tile: " << tile << " - time: " << mili << "ms";
		    	cout <<" nnz: " << TiledX[tile].totNnz << " nFibers: "
		    	<< TiledX[tile].fbrPtr[1].size() << " nSlc " << TiledX[tile].fbrIdx[0].size() << " ";
				cout << endl;
			} 
		} 
		allModeGPUTime += GPUTime; 
		cout << "singleCSF-GPU-mode " << MTTKRPmode <<" :" << GPUTime << "," << endl;
	}
	
	cout << "Total GPU time: " << allModeGPUTime << ", nnz:" << TiledX[0].totNnz 
		<< ", nFibers:" << TiledX[0].fbrPtr[1].size() << ", nSlc:" << TiledX[0].fbrIdx[0].size()
		<< endl;

	for (int bin = 0; bin < Opt.nBin; ++bin)
		hipStreamDestroy(streams[bin]);
	// check correctness
	if(Opt.impType == 14){
		MTTKRPmode = mode0;
		checkCuda(hipMemcpy(&U[MTTKRPmode].vals[0], dU0, U[MTTKRPmode].nRows * U[MTTKRPmode].nCols * sizeof(DTYPE), hipMemcpyDeviceToHost), 0);
	}
	else
		checkCuda(hipMemcpy(&U[mode0].vals[0], dU0, U[mode0].nRows * U[mode0].nCols * sizeof(DTYPE), hipMemcpyDeviceToHost), 0);
	hipFree(dVals); 
	hipFree(dU0); hipFree(dU1); hipFree(dU2); hipFree(dU3);
	hipFree(dfbrIdx0); hipFree(dInds2); hipFree(dInds3); 
	hipFree(dfbrIdx0); hipFree(dfbrIdx1); hipFree(dFbrIdx2);
	hipFree(dfbrPtr0); hipFree(dfbrPtr1); hipFree(dFbrPtr2);
	hipFree(dFbrLikeSlcInds);

	return 0;
}

int MTTKRP_HYB_GPU(const HYBTensor &HybX, Matrix *U, const Options &Opt){
	//allocate and memcpy GPU memory

	//Tensor
	ITYPE *dCOOInds0, *dCOOInds1, *dCOOInds2, *dCOOInds3;
	ITYPE *dCSLSlcPtr, *dCSLSlcInds, *dCSLInds1, *dCSLInds2, *dCSLSlcMapperBin;
	ITYPE *dfbrPtr0, *dfbrIdx0, *dInds2, *dInds3, *dfbrPtr1, *dfbrIdx1,  *dFbrPtr2, *dFbrIdx2, *dSlcMapperBin;

	DTYPE *dVals, *dCOOVals, *dCSLVals;
	ITYPE dLoc = 0, dSlcLoc = 0, dSlcIdxLoc = 0, dFbrLoc =0, dFbrIdxLoc =0, dBinLoc = 0, dCSLBinLoc = 0, dFbrLoc2 =0;
	int warpPerSlice = Opt.warpPerSlice;
	int logOfWarpPerSlice = log2(Opt.warpPerSlice);
	int TbPerSlc = 1;
	int logOfTPS = log2(TbPerSlc);

	// All tile same mode
	ITYPE mode0 = HybX.modeOrder[0];
    ITYPE mode1 = HybX.modeOrder[1];
    ITYPE mode2 = HybX.modeOrder[2];
    ITYPE mode3 =((HybX.ndims == 4) ? HybX.modeOrder[3] : 0) ;

    // ****** mem op HYB COO *******
    if(HybX.COOnnz > 0){
		
		checkCuda(hipMalloc((void**) &dCOOVals, HybX.COOnnz * sizeof(DTYPE)), 0);
		checkCuda(hipMalloc((void**) &dCOOInds0, HybX.COOnnz * sizeof(ITYPE)), 0);
		checkCuda(hipMalloc((void**) &dCOOInds1, HybX.COOnnz * sizeof(ITYPE)), 0);
		checkCuda(hipMalloc((void**) &dCOOInds2, HybX.COOnnz * sizeof(ITYPE)), 0);

		checkCuda(hipMemcpy(dCOOVals, &(HybX.COOvals[0]), HybX.COOnnz * sizeof(DTYPE),hipMemcpyHostToDevice), 0);
		checkCuda(hipMemcpy(dCOOInds0, &(HybX.COOinds[mode0][0]), HybX.COOnnz * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
		checkCuda(hipMemcpy(dCOOInds1, &(HybX.COOinds[mode1][0]), HybX.COOnnz * sizeof(ITYPE) ,hipMemcpyHostToDevice), 0);
		checkCuda(hipMemcpy(dCOOInds2, &(HybX.COOinds[mode2][0]), HybX.COOnnz * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
		
		if(HybX.ndims == 4){
			checkCuda(hipMalloc((void**) &dCOOInds3, HybX.COOnnz * sizeof(ITYPE)), 0);
			checkCuda(hipMemcpy(dCOOInds3, &(HybX.COOinds[mode3][0]), HybX.COOnnz * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
		}
	}

   // ****** mem op HYB CSL *******

	if(HybX.CSLnnz > 0){

		checkCuda(hipMalloc((void**) &dCSLVals, HybX.CSLnnz * sizeof(DTYPE)), 0);
		checkCuda(hipMalloc((void**) &dCSLSlcPtr,  HybX.CSLslicePtr.size() * sizeof(ITYPE)), 0);
		checkCuda(hipMalloc((void**) &dCSLSlcInds, HybX.CSLsliceIdx.size() * sizeof(ITYPE)), 0);
		checkCuda(hipMalloc((void**) &dCSLInds1, HybX.CSLnnz * sizeof(ITYPE)), 0);
		checkCuda(hipMalloc((void**) &dCSLInds2, HybX.CSLnnz * sizeof(ITYPE)), 0);
		checkCuda(hipMalloc((void**) &dCSLSlcMapperBin, HybX.CSLslicePtr.size() * sizeof(ITYPE)), 0);

		checkCuda(hipMemcpy(dCSLVals, &(HybX.CSLvals[0]), HybX.CSLnnz * sizeof(DTYPE),hipMemcpyHostToDevice), 0);	
		checkCuda(hipMemcpy(dCSLSlcPtr + dSlcLoc, &(HybX.CSLslicePtr[0]), HybX.CSLslicePtr.size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
		checkCuda(hipMemcpy(dCSLSlcInds + dSlcIdxLoc, &(HybX.CSLsliceIdx[0]), HybX.CSLsliceIdx.size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
		checkCuda(hipMemcpy(dCSLInds1, &(HybX.CSLinds[mode1][0]), HybX.CSLnnz * sizeof(ITYPE) ,hipMemcpyHostToDevice), 0);
		checkCuda(hipMemcpy(dCSLInds2, &(HybX.CSLinds[mode2][0]), HybX.CSLnnz * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
		
		dCSLBinLoc = 0;
		for (int bin = 0; bin < Opt.nBin; ++bin){

			if(bin > 0)
				dCSLBinLoc += HybX.CSLslcMapperBin[bin-1].size();

			if(HybX.CSLslcMapperBin[bin].size() > 0)
		    	checkCuda(hipMemcpy(dCSLSlcMapperBin + dSlcIdxLoc + dCSLBinLoc, &(HybX.CSLslcMapperBin[bin][0]), HybX.CSLslcMapperBin[bin].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
		}
	}
 
    // ****** mem op HYB HCSR *******

    if(HybX.HCSRnnz > 0){

		checkCuda(hipMalloc((void**) &dVals, HybX.HCSRnnz * sizeof(DTYPE)), 0);
		checkCuda(hipMalloc((void**) &dfbrPtr0,  HybX.fbrPtr[0].size() * sizeof(ITYPE)), 0);
		checkCuda(hipMalloc((void**) &dfbrIdx0, HybX.fbrIdx[0].size() * sizeof(ITYPE)), 0);
		checkCuda(hipMalloc((void**) &dSlcMapperBin, HybX.fbrPtr[0].size() * sizeof(ITYPE)), 0);
		checkCuda(hipMalloc((void**) &dfbrPtr1, HybX.fbrPtr[1].size()  * sizeof(ITYPE)), 0);
		checkCuda(hipMalloc((void**) &dfbrIdx1, HybX.fbrPtr[1].size() * sizeof(ITYPE)), 0);

		checkCuda(hipMemcpy(dVals, &(HybX.vals[0]), HybX.HCSRnnz * sizeof(DTYPE),hipMemcpyHostToDevice), 0);
		checkCuda(hipMemcpy(dfbrPtr0, &(HybX.fbrPtr[0][0]), HybX.fbrPtr[0].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
		checkCuda(hipMemcpy(dfbrIdx0, &(HybX.fbrIdx[0][0]), HybX.fbrIdx[0].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
		checkCuda(hipMemcpy(dfbrPtr1, &(HybX.fbrPtr[1][0]), HybX.fbrPtr[1].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
		checkCuda(hipMemcpy(dfbrIdx1, &(HybX.fbrIdx[1][0]), HybX.fbrPtr[1].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);

	    if(HybX.ndims == 3){
	    	checkCuda(hipMalloc((void**) &dInds2, HybX.HCSRnnz * sizeof(ITYPE)), 0);
	    	checkCuda(hipMemcpy(dInds2, &(HybX.inds[mode2][0]), HybX.HCSRnnz * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
		}

	    if(HybX.ndims == 4){
	        checkCuda(hipMalloc((void**) &dFbrIdx2, HybX.fbrIdx[2].size() * sizeof(ITYPE)), 0);
	        checkCuda(hipMalloc((void**) &dFbrPtr2, HybX.fbrPtr[2].size() * sizeof(ITYPE)), 0);
	        checkCuda(hipMalloc((void**) &dInds3, HybX.HCSRnnz * sizeof(ITYPE)), 0);
	        checkCuda(hipMemcpy(dFbrPtr2, &(HybX.fbrPtr[2][0]), HybX.fbrPtr[2].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
			checkCuda(hipMemcpy(dFbrIdx2, &(HybX.fbrIdx[2][0]), HybX.fbrIdx[2].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
			checkCuda(hipMemcpy(dInds3, &(HybX.inds[mode3][0]), HybX.HCSRnnz * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
	    }

		dBinLoc = 0;
		for (int bin = 0; bin < Opt.nBin; ++bin){

			if(bin > 0)
				dBinLoc += HybX.slcMapperBin[bin-1].size();

			if(HybX.slcMapperBin[bin].size() > 0)
		    	checkCuda(hipMemcpy(dSlcMapperBin + dSlcIdxLoc + dBinLoc, &(HybX.slcMapperBin[bin][0]), HybX.slcMapperBin[bin].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
		}
	}
	
	// //Matrices
	DTYPE *dU0, *dU1, *dU2, *dU3;	
	checkCuda(hipMalloc((void**) &dU0, U[mode0].nRows * U[mode0].nCols * sizeof(DTYPE)), 0);
	checkCuda(hipMalloc((void**) &dU1, U[mode1].nRows * U[mode1].nCols * sizeof(DTYPE)), 0);
	checkCuda(hipMalloc((void**) &dU2, U[mode2].nRows * U[mode2].nCols * sizeof(DTYPE)), 0);

	hipMemset(dU0, 0,  U[mode0].nRows * U[mode0].nCols * sizeof(DTYPE));
	checkCuda(hipMemcpy(dU1, &(U[mode1].vals[0]), U[mode1].nRows * U[mode1].nCols * sizeof(DTYPE), hipMemcpyHostToDevice), 0);
	checkCuda(hipMemcpy(dU2, &(U[mode2].vals[0]), U[mode2].nRows * U[mode2].nCols * sizeof(DTYPE), hipMemcpyHostToDevice), 0);

	if(HybX.ndims == 4){
        checkCuda(hipMalloc((void**) &dU3, U[mode3].nRows * U[mode3].nCols * sizeof(DTYPE)), 0);
        checkCuda(hipMemcpy(dU3, &(U[mode3].vals[0]), U[mode3].nRows * U[mode3].nCols * sizeof(DTYPE), hipMemcpyHostToDevice), 0);
    }
	
	// BLOCK and GRID
	int BLOCKSIZE = 512;
	dim3 block(BLOCKSIZE, 1, 1), grid(1, 1, 1);
	unsigned int rowInATB = BLOCKSIZE / (Opt.warpPerSlice*32); 

	if(Opt.warpPerSlice * 32 > BLOCKSIZE){
		cout << "BLOCKSIZE is smaller than work per slice! Increase BLOCKSIZE." << endl;
		exit(0);
	}

    hipEvent_t start, stop, HYBstart, HYBstop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventCreate(&HYBstart);
    hipEventCreate(&HYBstop);

    hipStream_t streams[2 * Opt.nBin + 1];
	for (int bin = 0; bin < 2 * Opt.nBin + 1; ++bin)
		hipStreamCreate(&streams[bin]);

    float mili = 0, HYBmili =0, GPUTime = 0, CPUtimer = 0, HYBTime = 0;
	dLoc = 0, dSlcLoc = 0, dSlcIdxLoc = 0; dFbrLoc =0;
	bool useLoop = false;
	
	if(useLoop)
		grid.x = 32768*2;

			// mili = 0; 
	dCSLBinLoc = 0; dBinLoc = 0;

	int smallBinEndsAt = 5;
	int slcPerTb = 0;

	cuda_timer_start(HYBstart);

	// ******* CUDA COO *******

	// if(HybX.COOnnz > 0){

	// 	BLOCKSIZE = 128;
	// 	block.x = BLOCKSIZE;
	// 		// /* Like PARTI loop */ = 

	// 	if(!useLoop)
	// 		grid.x = (32 * HybX.COOnnz + BLOCKSIZE - 1) / BLOCKSIZE;

	// 	if(Opt.verbose) 
	// 		cuda_timer_start(start);
  		
 //  		if(!useLoop){

	//   		if(HybX.ndims == 3)
	// 			mttkrp_HYB_COO_kernel<<<grid, block, 0, 0>>>(dCOOVals, dCOOInds0, dCOOInds1, dCOOInds2, HybX.COOnnz, dU0, dU1, dU2,	Opt.mode, Opt.R); 
	// 		else if (HybX.ndims == 4)
	// 			mttkrp_HYB_COO_kernel_4D<<<grid, block, 0, 0>>>(dCOOVals, dCOOInds0, dCOOInds1, dCOOInds2,dCOOInds3, HybX.COOnnz, dU0, dU1, dU2, dU3, Opt.mode, Opt.R); 
	// 	}

	// 	else{
  			
	//   		if(HybX.ndims == 3)
	// 			mttkrp_HYB_COO_kernel_loop<<<grid, block, 0, 0>>>(dCOOVals, dCOOInds0, dCOOInds1, dCOOInds2, HybX.COOnnz, dU0, dU1, dU2,	Opt.mode, Opt.R); 
	// 		else if (HybX.ndims == 4)
	// 			mttkrp_HYB_COO_kernel_4D_loop<<<grid, block, 0, 0>>>(dCOOVals, dCOOInds0, dCOOInds1, dCOOInds2,dCOOInds3, HybX.COOnnz, dU0, dU1, dU2, dU3, Opt.mode, Opt.R); 
	// 	}

	//     if(Opt.verbose){
	//     	cuda_timer_stop(start, stop, mili);
	//     	HYBTime += mili;
	//     	cout << "HYB-COO GPU " << mili << "ms"<< endl;
	//     }
	// }
	// ******* CUDA CSL *******

	// if(HybX.CSLnnz > 0 || HybX.HCSRnnz > 0)
	{
		if(HybX.COOnnz > 0){

			BLOCKSIZE = 128;
			block.x = 128;
			grid.x = (32 * HybX.COOnnz + BLOCKSIZE - 1) / BLOCKSIZE;

	  		if(HybX.ndims == 3)
				mttkrp_HYB_COO_kernel<<<grid, block, 0, 0>>>(dCOOVals, dCOOInds0, dCOOInds1, dCOOInds2, HybX.COOnnz, dU0, dU1, dU2,	Opt.mode, Opt.R); 
			else if (HybX.ndims == 4)
				mttkrp_HYB_COO_kernel_4D<<<grid, block, 0, 0>>>(dCOOVals, dCOOInds0, dCOOInds1, dCOOInds2,dCOOInds3, HybX.COOnnz, dU0, dU1, dU2, dU3, Opt.mode, Opt.R); 
		
		}

		BLOCKSIZE = 512;
		block.x = BLOCKSIZE;

		for (int bin = 0; bin < Opt.nBin ; ++bin){

			dBinLoc += ((bin > 0) ? HybX.slcMapperBin[bin-1].size() : 0);
			dCSLBinLoc += ((bin > 0) ? HybX.CSLslcMapperBin[bin-1].size() : 0);

			if( HybX.slcMapperBin[bin].size() == 0 && HybX.CSLslcMapperBin[bin].size() == 0)
				continue;
			// Processing small bin.. merged to one. 1 WARP slice
			if(bin < smallBinEndsAt){

				warpPerSlice = 1;
				logOfWarpPerSlice = 0;//log2(warpPerSlice);
				slcPerTb = 16 / warpPerSlice;

				/* CSL small bin */
				if(HybX.CSLnnz > 0){

					grid.x = ( warpPerSlice * 32 * HybX.CSLslcMapperBin[bin].size() + BLOCKSIZE - 1) / BLOCKSIZE;

					mttkrp_CSL_kernel_bin<<<grid, block, 0, streams[1]>>>(dCSLVals, dCSLSlcInds, dCSLSlcMapperBin + dCSLBinLoc, 
						dCSLInds2, dCSLSlcPtr, dCSLInds1, HybX.CSLslcMapperBin[bin].size(), 
						dU0, dU1, dU2, Opt.mode, Opt.R, warpPerSlice, logOfWarpPerSlice); 
				}
				
				/* HCSR small bin */
				if(HybX.HCSRnnz > 0){

					grid.x = ( warpPerSlice * 32 * HybX.slcMapperBin[bin].size() + BLOCKSIZE - 1) / BLOCKSIZE;

					if(HybX.ndims == 3)
						mttkrp_HCSR_kernel_smllBin<<<grid, block, 0, streams[2]>>>(dVals + dLoc, dfbrIdx0 + dSlcIdxLoc, dSlcMapperBin + dSlcIdxLoc + dBinLoc, 
						dInds2 + dLoc, dfbrPtr0 + dSlcLoc, dfbrPtr1 + dFbrLoc,  dfbrIdx1 + dFbrLoc, HybX.slcMapperBin[bin].size(), 
						dU0, dU1, dU2, Opt.mode, Opt.R, warpPerSlice, logOfWarpPerSlice, TbPerSlc, logOfTPS); 
					
					else if(HybX.ndims == 4)
						mttkrp_HCSR_kernel_smllBin_4D<<<grid, block, 0, streams[2]>>>(dVals + dLoc, dfbrIdx0 + dSlcIdxLoc, dSlcMapperBin + dSlcIdxLoc + dBinLoc, 
						dInds3 + dLoc, dfbrPtr0 + dSlcLoc, dfbrPtr1 + dFbrLoc,  dfbrIdx1 + dFbrLoc, dFbrPtr2 + dFbrLoc2, dFbrIdx2 + dFbrLoc2, HybX.slcMapperBin[bin].size(), 
						dU0, dU1, dU2, dU3, Opt.mode, Opt.R, warpPerSlice, logOfWarpPerSlice, TbPerSlc, logOfTPS); 
				}
			}

			// Processing heavy bin.. multiple TB per slice
			else{
		
				TbPerSlc = 1 << (bin - smallBinEndsAt + 1); // 1st big bin starts with 1 TB 1 << 1 not 1 << 5

				if(TbPerSlc > 32) TbPerSlc = 32;		
				logOfTPS = log2(TbPerSlc);

				warpPerSlice = 16;
				logOfWarpPerSlice = 4;

				/* CSL big bin */
				if(HybX.CSLnnz > 0){	
					grid.x = (TbPerSlc * warpPerSlice * 32 * HybX.CSLslcMapperBin[bin].size() + BLOCKSIZE - 1) / BLOCKSIZE;
					
					mttkrp_CSL_kernel_hvyBin<<<grid, block, 0, streams[bin+1]>>>(dCSLVals + dLoc, dCSLSlcInds + dSlcIdxLoc, dCSLSlcMapperBin + dSlcIdxLoc + dCSLBinLoc, 
						dCSLInds2 + dLoc, dCSLSlcPtr + dSlcLoc, dCSLInds1, HybX.CSLslcMapperBin[bin].size(), 
						dU0, dU1, dU2, Opt.mode, Opt.R, warpPerSlice, logOfWarpPerSlice,  TbPerSlc, logOfTPS); 
				}

				/* HCSR big bin */
				if(HybX.HCSRnnz > 0){
					grid.x = (TbPerSlc * warpPerSlice * 32 * HybX.slcMapperBin[bin].size() + BLOCKSIZE - 1) / BLOCKSIZE;
					
					if(HybX.ndims == 3)
						mttkrp_HCSR_kernel_hvyBin<<<grid, block, 0, streams[bin+2]>>>(dVals + dLoc, dfbrIdx0 + dSlcIdxLoc, dSlcMapperBin + dSlcIdxLoc + dBinLoc, 
							dInds2 + dLoc, dfbrPtr0 + dSlcLoc, dfbrPtr1 + dFbrLoc,  dfbrIdx1 + dFbrLoc, HybX.slcMapperBin[bin].size(), 
							dU0, dU1, dU2, Opt.mode, Opt.R, warpPerSlice, logOfWarpPerSlice,  TbPerSlc, logOfTPS); 
						
					else if(HybX.ndims == 4)
	                    mttkrp_HCSR_kernel_hvyBin_4D<<<grid, block, 0, streams[bin + 2]>>>(dVals + dLoc, dfbrIdx0 + dSlcIdxLoc, dSlcMapperBin + dSlcIdxLoc + dBinLoc, 
	                    dInds3 + dLoc, dfbrPtr0 + dSlcLoc, dfbrPtr1 + dFbrLoc, dfbrIdx1 + dFbrIdxLoc, dFbrPtr2 + dFbrLoc2, dFbrIdx2 + dFbrLoc2, HybX.slcMapperBin[bin].size(), 
	                    dU0, dU1, dU2, dU3, Opt.mode, Opt.R, warpPerSlice, logOfWarpPerSlice,  TbPerSlc, logOfTPS);
		        } 	

			}
		}

	    // if(Opt.verbose){
	    // 	cuda_timer_stop(start, stop, mili);
	    // 	HYBTime += mili;
	    // 	cout << "CSL+HCSR GPU-time: " << mili << "ms"<< endl;
	    // }
	}

	cuda_timer_stop(HYBstart, HYBstop, HYBmili);
	if(Opt.verbose)
		cout << "verbose on. HYB GPU: " << HYBmili << endl;
	else
		cout << "HYB GPU: " << HYBmili << endl;

	for (int bin = 0; bin < 2 * Opt.nBin + 1; ++bin)
		hipStreamDestroy(streams[bin]);
	// check correctness
	checkCuda(hipMemcpy(&U[mode0].vals[0], dU0, U[mode0].nRows * U[mode0].nCols * sizeof(DTYPE), hipMemcpyDeviceToHost), 0);
	hipFree(dVals); hipFree(dCOOVals); hipFree(dCSLVals); 
	hipFree(dU0); hipFree(dU1); hipFree(dU2);
	hipFree(dfbrIdx0); hipFree(dInds2); hipFree(dInds3); 
    hipFree(dfbrIdx0); hipFree(dfbrIdx1); hipFree(dFbrIdx2);
    hipFree(dfbrPtr0); hipFree(dfbrPtr1); hipFree(dFbrPtr2);
	hipFree(dCSLInds1); hipFree(dCSLInds2); hipFree(dCSLSlcPtr); hipFree(dCSLSlcInds);
	hipFree(dCOOInds0); hipFree(dCOOInds1); hipFree(dCOOInds2); 

	return 0;
}

int MTTKRP_MIHCSR_GPU(TiledTensor *TiledX, Matrix *U, const Options &Opt){

	ITYPE *dInds2, *dInds3, *dfbrPtr0, *dfbrIdx0, *dfbrPtr1, *dfbrIdx1, *dFbrPtr2, *dFbrIdx2, *dSlcMapperBin, *dFbrLikeSlcInds;
	DTYPE *dVals;
	ITYPE dLoc = 0, dSlcLoc = 0, dSlcIdxLoc = 0, dFbrLoc =0,  dFbrIdxLoc =0, dBinLoc = 0, dFbrLoc2 =0, dFbrLikeSlcIndsLoc = 0;
	ITYPE totNnz = 0, totSlcPtr = 0, totSlcIdx = 0, totFbrPtr = 0, totFbrIdx = 0, totFbrPtr2 = 0;

	// All m same mode
	ITYPE mode0 = 0;//TiledX[0].modeOrder[0];
    ITYPE mode1 = 1;;//TiledX[0].modeOrder[1];
    ITYPE mode2 = 2;//TiledX[0].modeOrder[2];
    ITYPE mode3 = 3;//((TiledX[0].ndims == 4) ? TiledX[0].modeOrder[3] : 0) ;

	for (int m = 0; m < TiledX[0].ndims; ++m){
		
		if (TiledX[m].totNnz == 0) continue;
		
		totNnz += TiledX[m].totNnz;
		totSlcPtr += TiledX[m].fbrPtr[0].size() ;
		totSlcIdx += TiledX[m].fbrIdx[0].size() ;
		totFbrPtr += TiledX[m].fbrPtr[1].size() ;
		totFbrIdx += TiledX[m].fbrIdx[1].size() ;
		totFbrPtr2 += ((TiledX[m].ndims == 4) ? TiledX[m].fbrPtr[2].size() : 0) ;

	}

	//allocate and memcpy GPU memory
	//Tensor
	checkCuda(hipMalloc((void**) &dVals, totNnz * sizeof(DTYPE)), 0);
	checkCuda(hipMalloc((void**) &dfbrPtr0, totSlcPtr * sizeof(ITYPE)), 0);
	checkCuda(hipMalloc((void**) &dfbrIdx0, totSlcIdx * sizeof(ITYPE)), 0);
	checkCuda(hipMalloc((void**) &dSlcMapperBin, totSlcPtr * sizeof(ITYPE)), 0);
	checkCuda(hipMalloc((void**) &dfbrPtr1, totFbrPtr * sizeof(ITYPE)), 0);
	checkCuda(hipMalloc((void**) &dfbrIdx1, totFbrIdx * sizeof(ITYPE)), 0);
	checkCuda(hipMalloc((void**) &dFbrLikeSlcInds, totFbrIdx * sizeof(ITYPE)), 0);

	if(TiledX[0].ndims == 3)
		checkCuda(hipMalloc((void**) &dInds2, totNnz * sizeof(ITYPE)), 0);

	if(TiledX[0].ndims == 4){
		checkCuda(hipMalloc((void**) &dFbrIdx2, totFbrPtr2 * sizeof(ITYPE)), 0);
		checkCuda(hipMalloc((void**) &dFbrPtr2, totFbrPtr2 * sizeof(ITYPE)), 0);
		checkCuda(hipMalloc((void**) &dInds3, totNnz * sizeof(ITYPE)), 0);
	}

	/* cuda memcopy for tiled parts*/
	for (int m = 0; m < TiledX[0].ndims; ++m){	

		if(m > 0) {

			if (TiledX[m-1].totNnz > 0) {
			
				dLoc += TiledX[m-1].totNnz;
				dSlcLoc += TiledX[m - 1].fbrPtr[0].size(); // all m same
				dSlcIdxLoc += TiledX[m - 1].fbrIdx[0].size(); 
				dFbrLoc += TiledX[m - 1].fbrPtr[1].size();
				dFbrIdxLoc += TiledX[m - 1].fbrIdx[1].size();
				dFbrLoc2 += ((TiledX[m].ndims == 4) ? TiledX[m - 1].fbrPtr[2].size() : 0) ;
				dFbrLikeSlcIndsLoc += TiledX[m - 1].fbrIdx[1].size();
			}
		}

		if (TiledX[m].totNnz == 0) continue;

		checkCuda(hipMemcpy(dVals + dLoc, &(TiledX[m].vals[0]), TiledX[m].totNnz * sizeof(DTYPE),hipMemcpyHostToDevice), 0);
		checkCuda(hipMemcpy(dfbrPtr0 + dSlcLoc, &(TiledX[m].fbrPtr[0][0]), TiledX[m].fbrPtr[0].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
		checkCuda(hipMemcpy(dfbrIdx0 + dSlcIdxLoc, &(TiledX[m].fbrIdx[0][0]), TiledX[m].fbrIdx[0].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
		checkCuda(hipMemcpy(dfbrPtr1 + dFbrLoc, &(TiledX[m].fbrPtr[1][0]), TiledX[m].fbrPtr[1].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
		checkCuda(hipMemcpy(dfbrIdx1 + dFbrIdxLoc, &(TiledX[m].fbrIdx[1][0]), TiledX[m].fbrIdx[1].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
		checkCuda(hipMemcpy(dFbrLikeSlcInds + dFbrIdxLoc, &(TiledX[m].fbrLikeSlcInds[0]), TiledX[m].fbrIdx[1].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);

		if(TiledX[m].ndims == 3){
			if(m == 0)
				// checkCuda(hipMemcpy(dInds2 + dLoc, &(TiledX[m].inds[mode2][0]), TiledX[m].totNnz * sizeof(ITYPE),hipMemcpyHostToDevice), 0);			
				checkCuda(hipMemcpy(dInds2 + dLoc, &(TiledX[m].inds[TiledX[m].modeOrder[2]][0]), TiledX[m].totNnz * sizeof(ITYPE),hipMemcpyHostToDevice), 0);			

			else if(m == 1)
				checkCuda(hipMemcpy(dInds2 + dLoc, &(TiledX[m].inds[TiledX[m].modeOrder[2]][0]), TiledX[m].totNnz * sizeof(ITYPE),hipMemcpyHostToDevice), 0);			
			else if(m == 2)
				checkCuda(hipMemcpy(dInds2 + dLoc, &(TiledX[m].inds[TiledX[m].modeOrder[2]][0]), TiledX[m].totNnz * sizeof(ITYPE),hipMemcpyHostToDevice), 0);			
		}
		if(TiledX[m].ndims == 4){			
			checkCuda(hipMemcpy(dFbrPtr2 + dFbrLoc2, &(TiledX[m].fbrPtr[2][0]), TiledX[m].fbrPtr[2].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
			checkCuda(hipMemcpy(dFbrIdx2 + dFbrLoc2, &(TiledX[m].fbrIdx[2][0]), TiledX[m].fbrIdx[2].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
			checkCuda(hipMemcpy(dInds3 + dLoc, &(TiledX[m].inds[mode3][0]), TiledX[m].totNnz * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
		}

		dBinLoc = 0;
		for (int bin = 0; bin < Opt.nBin; ++bin){

			if(bin > 0)
				dBinLoc += TiledX[m].slcMapperBin[bin-1].size();

		    checkCuda(hipMemcpy(dSlcMapperBin + dSlcIdxLoc + dBinLoc, &(TiledX[m].slcMapperBin[bin][0]), TiledX[m].slcMapperBin[bin].size() * sizeof(ITYPE),hipMemcpyHostToDevice), 0);
		}
	}

	// //Matrices
	DTYPE *dU0, *dU1, *dU2, *dU3;	
	checkCuda(hipMalloc((void**) &dU0, U[mode0].nRows * U[mode0].nCols * sizeof(DTYPE)), 0);
	checkCuda(hipMalloc((void**) &dU1, U[mode1].nRows * U[mode1].nCols * sizeof(DTYPE)), 0);
	checkCuda(hipMalloc((void**) &dU2, U[mode2].nRows * U[mode2].nCols * sizeof(DTYPE)), 0);

	hipMemset(dU0, 0,  U[mode0].nRows * U[mode0].nCols * sizeof(DTYPE));
	checkCuda(hipMemcpy(dU1, &(U[mode1].vals[0]), U[mode1].nRows * U[mode1].nCols * sizeof(DTYPE), hipMemcpyHostToDevice), 0);
	checkCuda(hipMemcpy(dU2, &(U[mode2].vals[0]), U[mode2].nRows * U[mode2].nCols * sizeof(DTYPE), hipMemcpyHostToDevice), 0);
	
	if(TiledX[0].ndims == 4){
		checkCuda(hipMalloc((void**) &dU3, U[mode3].nRows * U[mode3].nCols * sizeof(DTYPE)), 0);
		checkCuda(hipMemcpy(dU3, &(U[mode3].vals[0]), U[mode3].nRows * U[mode3].nCols * sizeof(DTYPE), hipMemcpyHostToDevice), 0);
	}

	// BLOCK and GRID
	int BLOCKSIZE = 512;
	unsigned int rowInATB = BLOCKSIZE / (Opt.warpPerSlice*32); 

	if(Opt.warpPerSlice * 32 > BLOCKSIZE){
		cout << "BLOCKSIZE is smaller than work per slice! Increase BLOCKSIZE." << endl;
		exit(0);
	}

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipStream_t streams[Opt.nBin];
    float mili = 0, GPUTime = 0, CPUtimer = 0, allModeGPUTime = 0;

    int smallBinEndsAt = 5;

    /* Warp per slice and threadblock per slice */
    int *warpPerSlc = new int[Opt.nBin];
    int *logOfWarpPerSlc = new int[Opt.nBin];
    int *TbPerSlc = new int[Opt.nBin];
    int *logOfTbPerSlc = new int[Opt.nBin];

    for (int bin = 0; bin < Opt.nBin ; ++bin){
    	
    	TbPerSlc[bin] = 1;
		warpPerSlc[bin] = ((bin > 0) ? 2 << (bin - 1) : 1);
		
		if(warpPerSlc[bin] > 16)		
			warpPerSlc[bin] = 16;

		logOfWarpPerSlc[bin] = log2(warpPerSlc[bin]);

		TbPerSlc[bin] = 1;
		logOfTbPerSlc[bin] = 0;
		
		if (bin >= smallBinEndsAt){
		
			TbPerSlc[bin] = 1 << (bin - smallBinEndsAt + 1); // 1st big bin starts with 1 TB 1 << 1 not 1 << 5
			if(TbPerSlc[bin] > 32) TbPerSlc[bin] = 32;		
			logOfTbPerSlc[bin] = log2(TbPerSlc[bin]);

			warpPerSlc[bin] = 16;
			logOfWarpPerSlc[bin] = 4;
		}
    }

    // TBD: change warpPerSlc to warpPerSlc[bin] and all
	int slcPerTb = 1;

	dLoc = 0, dSlcLoc = 0, dSlcIdxLoc = 0; dFbrLoc =0, dFbrIdxLoc = 0, dFbrLoc2= 0;

	for (int bin = 0; bin < Opt.nBin; ++bin)
		hipStreamCreate(&streams[bin]);

	/* MTTKRP on mode 0,1,2 using MICSF*/
	cout << "change loop, double!!" << endl;

	for (int MTTKRPmode = 0; MTTKRPmode < TiledX[0].ndims; ++MTTKRPmode){

		if(MTTKRPmode > 0){

			mili = 0; GPUTime = 0; CPUtimer = 0;
			dLoc = 0; dSlcLoc = 0; dSlcIdxLoc = 0; dFbrLoc =0; dFbrIdxLoc = 0; dFbrLoc2= 0, dFbrLikeSlcIndsLoc = 0;

			// MTTKRP on mode mode 0 changed DU0. To pass correctness for now initializing to 2 again.
			int mode = MTTKRPmode - 1;
		    for(long r = 0; r < U[mode].nRows; ++r){
		        for(long c = 0; c < U[mode].nCols; ++c) // or u[mode].nCols 
		            U[mode].vals[r * U[mode].nCols + c] = mode + .5;//0.1 * drand48(); //1 ;//(r * R + c + 1); //
		    }

		    if(MTTKRPmode == 1){
		    	checkCuda(hipMemcpy(dU0, &(U[mode0].vals[0]), U[mode0].nRows * U[mode0].nCols * sizeof(DTYPE), hipMemcpyHostToDevice), 0);	
				hipMemset(dU1, 0,  U[mode1].nRows * U[mode1].nCols * sizeof(DTYPE));
			}
			else if(MTTKRPmode == 2){
				// checkCuda(hipMemcpy(dU0, &(U[mode0].vals[0]), U[mode0].nRows * U[mode0].nCols * sizeof(DTYPE), hipMemcpyHostToDevice), 0);	
				checkCuda(hipMemcpy(dU1, &(U[mode1].vals[0]), U[mode1].nRows * U[mode1].nCols * sizeof(DTYPE), hipMemcpyHostToDevice), 0);	
				hipMemset(dU2, 0,  U[mode2].nRows * U[mode2].nCols * sizeof(DTYPE));
			}

		}
		
		for (int m = 0; m < TiledX[0].ndims; ++m){
	
			int orgMode = (TiledX[m].modeOrder[0] + 2) % TiledX[m].ndims;

			dBinLoc = 0;
			
			if(m > 0) {

				if (TiledX[m-1].totNnz > 0) {

					dLoc += TiledX[m-1].totNnz;
					dSlcLoc += TiledX[m - 1].fbrPtr[0].size(); 
					dSlcIdxLoc += TiledX[m - 1].fbrIdx[0].size(); 
					dFbrLoc += TiledX[m - 1].fbrPtr[1].size();
					dFbrIdxLoc += TiledX[m - 1].fbrIdx[1].size();
					dFbrLoc2 += ((TiledX[0].ndims == 4) ? TiledX[m - 1].fbrPtr[2].size() : 0) ;
					dFbrLikeSlcIndsLoc += TiledX[m - 1].fbrIdx[1].size();
				}
			}

			BLOCKSIZE = 512;
			dim3 block(BLOCKSIZE, 1, 1), grid(1, 1, 1);

			if (TiledX[m].totNnz == 0) continue;

			// Process small bins.. accepts 2 slice 1 TB

			cuda_timer_start(start);

			if(TiledX[m].modeOrder[0] == MTTKRPmode && TiledX[m].totNnz){
				
				if(Opt.verbose)
					cout << "Slc atomics - " ;

				BLOCKSIZE = 128;
				dim3 block(BLOCKSIZE, 1, 1), grid(1, 1, 1);
				int slcPerTb = 0;
				int warpPerFbr = 1;//BLOCKSIZE/32;//1;//Opt.warpPerSlice;//4;//;
				int logOfWarpPerFbr = log2(warpPerFbr);
				int bin = 0;
				bool useLoop = false;
				int fbrPerWarp = 4; // dont overflow TB
				int logOfFbrPerWarp = log2(fbrPerWarp );

	
				grid.x = ( warpPerFbr * 32 * ((TiledX[m].nFibers + fbrPerWarp-1)/fbrPerWarp) + BLOCKSIZE - 1) / BLOCKSIZE;
						
				if(MTTKRPmode == 0){
				
					if(TiledX[m].modeOrder[2] == orgMode)
						mttkrp_MIHCSR_kernel_slc_atomic_fbrLvlPar<<<grid, block, 0, streams[bin]>>>(dVals + dLoc, dFbrLikeSlcInds + dFbrLoc, 
						dInds2 + dLoc, dfbrPtr0 + dSlcLoc, dfbrPtr1 + dFbrLoc, dfbrIdx1 + dFbrLoc, TiledX[m].nFibers, 
						dU0, dU1, dU2, Opt.mode, Opt.R, warpPerFbr, logOfWarpPerFbr, fbrPerWarp, logOfFbrPerWarp);

					else
						mttkrp_MIHCSR_kernel_slc_atomic_fbrLvlPar<<<grid, block, 0, streams[bin]>>>(dVals + dLoc, dFbrLikeSlcInds + dFbrLoc, 
						dInds2 + dLoc, dfbrPtr0 + dSlcLoc, dfbrPtr1 + dFbrLoc, dfbrIdx1 + dFbrLoc, TiledX[m].nFibers, 
						dU0, dU2, dU1, Opt.mode, Opt.R, warpPerFbr, logOfWarpPerFbr, fbrPerWarp, logOfFbrPerWarp);
				}
				else if(MTTKRPmode == 1){
				
					if(TiledX[m].modeOrder[2] == orgMode)
						mttkrp_MIHCSR_kernel_slc_atomic_fbrLvlPar<<<grid, block, 0, streams[bin]>>>(dVals + dLoc, dFbrLikeSlcInds + dFbrLoc, 
						dInds2 + dLoc, dfbrPtr0 + dSlcLoc, dfbrPtr1 + dFbrLoc, dfbrIdx1 + dFbrLoc, TiledX[m].nFibers, 
						dU1, dU2, dU0, Opt.mode, Opt.R, warpPerFbr, logOfWarpPerFbr, fbrPerWarp, logOfFbrPerWarp);
					else
						mttkrp_MIHCSR_kernel_slc_atomic_fbrLvlPar<<<grid, block, 0, streams[bin]>>>(dVals + dLoc, dFbrLikeSlcInds + dFbrLoc, 
						dInds2 + dLoc, dfbrPtr0 + dSlcLoc, dfbrPtr1 + dFbrLoc, dfbrIdx1 + dFbrLoc, TiledX[m].nFibers, 
						dU1, dU0, dU2, Opt.mode, Opt.R, warpPerFbr, logOfWarpPerFbr, fbrPerWarp, logOfFbrPerWarp);
				}
				
				else if(MTTKRPmode == 2){
				
					if(TiledX[m].modeOrder[2] == orgMode)
						mttkrp_MIHCSR_kernel_slc_atomic_fbrLvlPar<<<grid, block, 0, streams[bin]>>>(dVals + dLoc, dFbrLikeSlcInds + dFbrLoc, 
						dInds2 + dLoc, dfbrPtr0 + dSlcLoc, dfbrPtr1 + dFbrLoc, dfbrIdx1 + dFbrLoc, TiledX[m].nFibers, 
						dU2, dU0, dU1, Opt.mode, Opt.R, warpPerFbr, logOfWarpPerFbr, fbrPerWarp, logOfFbrPerWarp);
					else
						mttkrp_MIHCSR_kernel_slc_atomic_fbrLvlPar<<<grid, block, 0, streams[bin]>>>(dVals + dLoc, dFbrLikeSlcInds + dFbrLoc, 
						dInds2 + dLoc, dfbrPtr0 + dSlcLoc, dfbrPtr1 + dFbrLoc, dfbrIdx1 + dFbrLoc, TiledX[m].nFibers, 
						dU2, dU1, dU0, Opt.mode, Opt.R, warpPerFbr, logOfWarpPerFbr, fbrPerWarp, logOfFbrPerWarp);
				}
				{
			
					// for (int bin = 0; bin < Opt.nBin ; ++bin){

					// 	if(bin < smallBinEndsAt){

					// 		dBinLoc += ((bin > 0) ? TiledX[m].slcMapperBin[bin-1].size() : 0);

					// 		grid.x = ( TbPerSlc[bin] * warpPerSlc[bin] * 32 * TiledX[m].slcMapperBin[bin].size() + BLOCKSIZE - 1) / BLOCKSIZE;
							
					// 		//only diff order between du0, du1,du2
					// 		if(MTTKRPmode == 0){
							
					// 			if(TiledX[m].modeOrder[2] == orgMode)
					// 				mttkrp_HCSR_kernel_smllBin<<<grid, block, 0 , streams[bin]>>>(dVals + dLoc, dfbrIdx0 + dSlcIdxLoc, dSlcMapperBin + dSlcIdxLoc + dBinLoc, 
					// 				dInds2 + dLoc, dfbrPtr0 + dSlcLoc, dfbrPtr1 + dFbrLoc,  dfbrIdx1 + dFbrLoc, TiledX[m].slcMapperBin[bin].size(), 
					// 				dU0, dU1, dU2, Opt.mode, Opt.R, warpPerSlc[bin], logOfWarpPerSlc[bin], TbPerSlc[bin], logOfTbPerSlc[bin]); 
					// 			else
					// 				mttkrp_HCSR_kernel_smllBin<<<grid, block, 0 , streams[bin]>>>(dVals + dLoc, dfbrIdx0 + dSlcIdxLoc, dSlcMapperBin + dSlcIdxLoc + dBinLoc, 
					// 				dInds2 + dLoc, dfbrPtr0 + dSlcLoc, dfbrPtr1 + dFbrLoc,  dfbrIdx1 + dFbrLoc, TiledX[m].slcMapperBin[bin].size(), 
					// 				dU0, dU2, dU1, Opt.mode, Opt.R, warpPerSlc[bin], logOfWarpPerSlc[bin], TbPerSlc[bin], logOfTbPerSlc[bin]); 
					// 		}
					// 		else if(MTTKRPmode == 1){

					// 			if(TiledX[m].modeOrder[2] == orgMode)
					// 				mttkrp_HCSR_kernel_smllBin<<<grid, block, 0 , streams[bin]>>>(dVals + dLoc, dfbrIdx0 + dSlcIdxLoc, dSlcMapperBin + dSlcIdxLoc + dBinLoc, 
					// 				dInds2 + dLoc, dfbrPtr0 + dSlcLoc, dfbrPtr1 + dFbrLoc,  dfbrIdx1 + dFbrLoc, TiledX[m].slcMapperBin[bin].size(), 
					// 				dU1, dU2, dU0, Opt.mode, Opt.R, warpPerSlc[bin], logOfWarpPerSlc[bin], TbPerSlc[bin], logOfTbPerSlc[bin]); 
					// 			else
					// 				mttkrp_HCSR_kernel_smllBin<<<grid, block, 0 , streams[bin]>>>(dVals + dLoc, dfbrIdx0 + dSlcIdxLoc, dSlcMapperBin + dSlcIdxLoc + dBinLoc, 
					// 				dInds2 + dLoc, dfbrPtr0 + dSlcLoc, dfbrPtr1 + dFbrLoc,  dfbrIdx1 + dFbrLoc, TiledX[m].slcMapperBin[bin].size(), 
					// 				dU1, dU0, dU2, Opt.mode, Opt.R, warpPerSlc[bin], logOfWarpPerSlc[bin], TbPerSlc[bin], logOfTbPerSlc[bin]); 

					// 		}
					// 		else if(MTTKRPmode == 2){

					// 			if(TiledX[m].modeOrder[2] == orgMode)
					// 				mttkrp_HCSR_kernel_smllBin<<<grid, block, 0 , streams[bin]>>>(dVals + dLoc, dfbrIdx0 + dSlcIdxLoc, dSlcMapperBin + dSlcIdxLoc + dBinLoc, 
					// 				dInds2 + dLoc, dfbrPtr0 + dSlcLoc, dfbrPtr1 + dFbrLoc,  dfbrIdx1 + dFbrLoc, TiledX[m].slcMapperBin[bin].size(), 
					// 				dU2, dU0, dU1, Opt.mode, Opt.R, warpPerSlc[bin], logOfWarpPerSlc[bin], TbPerSlc[bin], logOfTbPerSlc[bin]); 
					// 			else
					// 				mttkrp_HCSR_kernel_smllBin<<<grid, block, 0 , streams[bin]>>>(dVals + dLoc, dfbrIdx0 + dSlcIdxLoc, dSlcMapperBin + dSlcIdxLoc + dBinLoc, 
					// 				dInds2 + dLoc, dfbrPtr0 + dSlcLoc, dfbrPtr1 + dFbrLoc,  dfbrIdx1 + dFbrLoc, TiledX[m].slcMapperBin[bin].size(), 
					// 				dU2, dU1, dU0, Opt.mode, Opt.R, warpPerSlc[bin], logOfWarpPerSlc[bin], TbPerSlc[bin], logOfTbPerSlc[bin]); 
					// 			}
								
					// 	}

						
					// 	// Processing heavy bin.. multiple TB per slice
					// 	else{

					// 		dBinLoc += TiledX[m].slcMapperBin[bin-1].size();
									
					// 		grid.x = (TbPerSlc[bin] * warpPerSlc[bin] * 32 * TiledX[m].slcMapperBin[bin].size() + BLOCKSIZE - 1) / BLOCKSIZE;
									
					// 		if(MTTKRPmode == 0){		
							
					// 			if(TiledX[m].modeOrder[2] == orgMode)
					// 				mttkrp_HCSR_kernel_hvyBin<<<grid, block, 0, streams[bin]>>>(dVals + dLoc, dfbrIdx0 + dSlcIdxLoc, dSlcMapperBin + dSlcIdxLoc + dBinLoc, 
					// 				dInds2 + dLoc, dfbrPtr0 + dSlcLoc, dfbrPtr1 + dFbrLoc,  dfbrIdx1 + dFbrLoc, TiledX[m].slcMapperBin[bin].size(), 
					// 				dU0, dU1, dU2, Opt.mode, Opt.R, warpPerSlc[bin], logOfWarpPerSlc[bin],  TbPerSlc[bin], logOfTbPerSlc[bin]); 
					// 			else
					// 				mttkrp_HCSR_kernel_hvyBin<<<grid, block, 0, streams[bin]>>>(dVals + dLoc, dfbrIdx0 + dSlcIdxLoc, dSlcMapperBin + dSlcIdxLoc + dBinLoc, 
					// 				dInds2 + dLoc, dfbrPtr0 + dSlcLoc, dfbrPtr1 + dFbrLoc,  dfbrIdx1 + dFbrLoc, TiledX[m].slcMapperBin[bin].size(), 
					// 				dU0, dU2, dU1, Opt.mode, Opt.R, warpPerSlc[bin], logOfWarpPerSlc[bin],  TbPerSlc[bin], logOfTbPerSlc[bin]); 
					// 		}

					// 		else if(MTTKRPmode == 1){		
							
					// 			if(TiledX[m].modeOrder[2] == orgMode)
					// 				mttkrp_HCSR_kernel_hvyBin<<<grid, block, 0, streams[bin]>>>(dVals + dLoc, dfbrIdx0 + dSlcIdxLoc, dSlcMapperBin + dSlcIdxLoc + dBinLoc, 
					// 				dInds2 + dLoc, dfbrPtr0 + dSlcLoc, dfbrPtr1 + dFbrLoc,  dfbrIdx1 + dFbrLoc, TiledX[m].slcMapperBin[bin].size(), 
					// 				dU1, dU2, dU0, Opt.mode, Opt.R, warpPerSlc[bin], logOfWarpPerSlc[bin],  TbPerSlc[bin], logOfTbPerSlc[bin]); 
					// 			else
					// 				mttkrp_HCSR_kernel_hvyBin<<<grid, block, 0, streams[bin]>>>(dVals + dLoc, dfbrIdx0 + dSlcIdxLoc, dSlcMapperBin + dSlcIdxLoc + dBinLoc, 
					// 				dInds2 + dLoc, dfbrPtr0 + dSlcLoc, dfbrPtr1 + dFbrLoc,  dfbrIdx1 + dFbrLoc, TiledX[m].slcMapperBin[bin].size(), 
					// 				dU1, dU0, dU2, Opt.mode, Opt.R, warpPerSlc[bin], logOfWarpPerSlc[bin],  TbPerSlc[bin], logOfTbPerSlc[bin]); 
					// 		}


					// 		else if(MTTKRPmode == 2){		
							
					// 			if(TiledX[m].modeOrder[2] == orgMode)
					// 				mttkrp_HCSR_kernel_hvyBin<<<grid, block, 0, streams[bin]>>>(dVals + dLoc, dfbrIdx0 + dSlcIdxLoc, dSlcMapperBin + dSlcIdxLoc + dBinLoc, 
					// 				dInds2 + dLoc, dfbrPtr0 + dSlcLoc, dfbrPtr1 + dFbrLoc,  dfbrIdx1 + dFbrLoc, TiledX[m].slcMapperBin[bin].size(), 
					// 				dU2, dU0, dU1, Opt.mode, Opt.R, warpPerSlc[bin], logOfWarpPerSlc[bin],  TbPerSlc[bin], logOfTbPerSlc[bin]); 
					// 			else
					// 				mttkrp_HCSR_kernel_hvyBin<<<grid, block, 0, streams[bin]>>>(dVals + dLoc, dfbrIdx0 + dSlcIdxLoc, dSlcMapperBin + dSlcIdxLoc + dBinLoc, 
					// 				dInds2 + dLoc, dfbrPtr0 + dSlcLoc, dfbrPtr1 + dFbrLoc,  dfbrIdx1 + dFbrLoc, TiledX[m].slcMapperBin[bin].size(), 
					// 				dU2, dU1, dU0, Opt.mode, Opt.R, warpPerSlc[bin], logOfWarpPerSlc[bin],  TbPerSlc[bin], logOfTbPerSlc[bin]); 
					// 		}
					// 	}
					// }
				}
			}

			else if(TiledX[m].modeOrder[1] == MTTKRPmode && TiledX[m].totNnz){
			
				if(Opt.verbose)
					cout << "Fbr atomics - " ;

				BLOCKSIZE = 128;
				dim3 block(BLOCKSIZE, 1, 1), grid(1, 1, 1);

				int warpPerFbr = 1;//Opt.warpPerSlice;//4;//;BLOCKSIZE/32;//
				int logOfWarpPerFbr = log2(warpPerFbr);
				int bin = 0;
				
				grid.x = ( warpPerFbr * 32 * TiledX[m].nFibers + BLOCKSIZE - 1) / BLOCKSIZE;
						
				if(MTTKRPmode == 0){
				
					if(TiledX[m].modeOrder[2] == orgMode)
						mttkrp_MIHCSR_kernel_fbr_atomic_fbrLvlPar<<<grid, block, 0, streams[bin]>>>(dVals + dLoc, dFbrLikeSlcInds + dFbrLoc, 
						dInds2 + dLoc, dfbrPtr0 + dSlcLoc, dfbrPtr1 + dFbrLoc,  dfbrIdx1 + dFbrLoc, TiledX[m].nFibers, 
						dU0, dU1, dU2, Opt.mode, Opt.R, warpPerFbr, logOfWarpPerFbr);

					else
						mttkrp_MIHCSR_kernel_fbr_atomic_fbrLvlPar<<<grid, block, 0, streams[bin]>>>(dVals + dLoc, dFbrLikeSlcInds + dFbrLoc, 
						dInds2 + dLoc, dfbrPtr0 + dSlcLoc, dfbrPtr1 + dFbrLoc,  dfbrIdx1 + dFbrLoc, TiledX[m].nFibers, 
						dU0, dU2, dU1, Opt.mode, Opt.R, warpPerFbr, logOfWarpPerFbr);
				}
				else if(MTTKRPmode == 1){
				
					if(TiledX[m].modeOrder[2] == orgMode)
						mttkrp_MIHCSR_kernel_fbr_atomic_fbrLvlPar<<<grid, block, 0, streams[bin]>>>(dVals + dLoc, dFbrLikeSlcInds + dFbrLoc, 
						dInds2 + dLoc, dfbrPtr0 + dSlcLoc, dfbrPtr1 + dFbrLoc,  dfbrIdx1 + dFbrLoc, TiledX[m].nFibers, 
						dU1, dU2, dU0, Opt.mode, Opt.R, warpPerFbr, logOfWarpPerFbr);
					else
						mttkrp_MIHCSR_kernel_fbr_atomic_fbrLvlPar<<<grid, block, 0, streams[bin]>>>(dVals + dLoc, dFbrLikeSlcInds + dFbrLoc, 
						dInds2 + dLoc, dfbrPtr0 + dSlcLoc, dfbrPtr1 + dFbrLoc,  dfbrIdx1 + dFbrLoc, TiledX[m].nFibers, 
						dU1, dU0, dU2, Opt.mode, Opt.R, warpPerFbr, logOfWarpPerFbr);
				}
				
				else if(MTTKRPmode == 2){
				
					if(TiledX[m].modeOrder[2] == orgMode)
						mttkrp_MIHCSR_kernel_fbr_atomic_fbrLvlPar<<<grid, block, 0, streams[bin]>>>(dVals + dLoc, dFbrLikeSlcInds + dFbrLoc, 
						dInds2 + dLoc, dfbrPtr0 + dSlcLoc, dfbrPtr1 + dFbrLoc,  dfbrIdx1 + dFbrLoc, TiledX[m].nFibers, 
						dU2, dU0, dU1, Opt.mode, Opt.R, warpPerFbr, logOfWarpPerFbr);
					else
						mttkrp_MIHCSR_kernel_fbr_atomic_fbrLvlPar<<<grid, block, 0, streams[bin]>>>(dVals + dLoc, dFbrLikeSlcInds + dFbrLoc, 
						dInds2 + dLoc, dfbrPtr0 + dSlcLoc, dfbrPtr1 + dFbrLoc,  dfbrIdx1 + dFbrLoc, TiledX[m].nFibers, 
						dU2, dU1, dU0, Opt.mode, Opt.R, warpPerFbr, logOfWarpPerFbr);
				}
			}

			else if(TiledX[m].modeOrder[2] == MTTKRPmode && TiledX[m].totNnz){

				if(Opt.verbose)
					cout << "nnz atomics - " ;

				BLOCKSIZE = 128;
				dim3 block(BLOCKSIZE, 1, 1), grid(1, 1, 1);

				int warpPerFbr = 1;//Opt.warpPerSlice;//4;//;BLOCKSIZE/32;//
				int logOfWarpPerFbr = log2(warpPerFbr);
				int bin = 0;
				
				grid.x = ( warpPerFbr * 32 * TiledX[m].nFibers + BLOCKSIZE - 1) / BLOCKSIZE;
						
				if(MTTKRPmode == 0){
				
					if(TiledX[m].modeOrder[2] == orgMode)
					    mttkrp_MIHCSR_kernel_all_atomic_fbrLvlPar<<<grid, block, 0, streams[bin]>>>(dVals + dLoc, dFbrLikeSlcInds + dFbrLoc, 
						dInds2 + dLoc, dfbrPtr0 + dSlcLoc, dfbrPtr1 + dFbrLoc,  dfbrIdx1 + dFbrLoc, TiledX[m].nFibers, 
						dU0, dU1, dU2, Opt.mode, Opt.R, warpPerFbr, logOfWarpPerFbr); 

					else
						mttkrp_MIHCSR_kernel_all_atomic_fbrLvlPar<<<grid, block, 0, streams[bin]>>>(dVals + dLoc, dFbrLikeSlcInds + dFbrLoc, 
						dInds2 + dLoc, dfbrPtr0 + dSlcLoc, dfbrPtr1 + dFbrLoc,  dfbrIdx1 + dFbrLoc, TiledX[m].nFibers, 
						dU0, dU2, dU1, Opt.mode, Opt.R, warpPerFbr, logOfWarpPerFbr);
				}
				else if(MTTKRPmode == 1){
				
					if(TiledX[m].modeOrder[2] == orgMode)
						mttkrp_MIHCSR_kernel_all_atomic_fbrLvlPar<<<grid, block, 0, streams[bin]>>>(dVals + dLoc, dFbrLikeSlcInds + dFbrLoc, 
						dInds2 + dLoc, dfbrPtr0 + dSlcLoc, dfbrPtr1 + dFbrLoc,  dfbrIdx1 + dFbrLoc, TiledX[m].nFibers, 
						dU1, dU2, dU0, Opt.mode, Opt.R, warpPerFbr, logOfWarpPerFbr);
					else
						mttkrp_MIHCSR_kernel_all_atomic_fbrLvlPar<<<grid, block, 0, streams[bin]>>>(dVals + dLoc, dFbrLikeSlcInds + dFbrLoc, 
						dInds2 + dLoc, dfbrPtr0 + dSlcLoc, dfbrPtr1 + dFbrLoc,  dfbrIdx1 + dFbrLoc, TiledX[m].nFibers, 
						dU1, dU0, dU2, Opt.mode, Opt.R, warpPerFbr, logOfWarpPerFbr);
				}
				
				else if(MTTKRPmode == 2){
				
					if(TiledX[m].modeOrder[2] == orgMode)
						mttkrp_MIHCSR_kernel_all_atomic_fbrLvlPar<<<grid, block, 0, streams[bin]>>>(dVals + dLoc, dFbrLikeSlcInds + dFbrLoc, 
						dInds2 + dLoc, dfbrPtr0 + dSlcLoc, dfbrPtr1 + dFbrLoc,  dfbrIdx1 + dFbrLoc, TiledX[m].nFibers, 
						dU2, dU0, dU1, Opt.mode, Opt.R, warpPerFbr, logOfWarpPerFbr);
					else
						mttkrp_MIHCSR_kernel_all_atomic_fbrLvlPar<<<grid, block, 0, streams[bin]>>>(dVals + dLoc, dFbrLikeSlcInds + dFbrLoc, 
						dInds2 + dLoc, dfbrPtr0 + dSlcLoc, dfbrPtr1 + dFbrLoc,  dfbrIdx1 + dFbrLoc, TiledX[m].nFibers, 
						dU2, dU1, dU0, Opt.mode, Opt.R, warpPerFbr, logOfWarpPerFbr);
				}
			}
		
			cuda_timer_stop(start, stop, mili);
		    GPUTime += mili;

		    if(Opt.verbose)
		    {
		    	cout << "Tile: " << m << " - time: " << mili << " ms";
		    	cout <<" nnz: " << TiledX[m].totNnz << " nFibers: "
		    	<< TiledX[m].fbrPtr[1].size() << " nSlc " << TiledX[m].fbrIdx[0].size() << " ";
				cout << " modeOrder: " << TiledX[m].modeOrder[0] <<" " << TiledX[m].modeOrder[1] <<" "
				<< TiledX[m].modeOrder[2];
				cout << endl;
			}   
		}
		cout << "MI-HCSR-GPU-mode "<< MTTKRPmode <<" : " << GPUTime << "," << endl;
		allModeGPUTime += GPUTime; 
	}
	int totalMIslics = 0, totalMIfibers = 0, totalMInnz = 0;;
	for (int m = 0; m <  TiledX[0].ndims; ++m){
		if(TiledX[m].totNnz){
			totalMIslics += TiledX[m].fbrIdx[0].size();
			totalMIfibers += TiledX[m].fbrPtr[1].size();
			totalMInnz += TiledX[m].totNnz;
		}
	}

	cout << "Total GPU time: " << allModeGPUTime << ", nnz:" << totalMInnz 
			<< ", nFibers:" << totalMIfibers << ", nSlc:" << totalMIslics 
			<< endl;

	for (int bin = 0; bin < Opt.nBin; ++bin)
		hipStreamDestroy(streams[bin]);

	// Copying output matrix from GPU to CPU
	
	if(Opt.impType == 12){
		int MTTKRPmode = mode2;
		checkCuda(hipMemcpy(&U[MTTKRPmode].vals[0], dU2, U[MTTKRPmode].nRows * U[MTTKRPmode].nCols * sizeof(DTYPE), hipMemcpyDeviceToHost), 0);
	}
	else
		checkCuda(hipMemcpy(&U[Opt.mode].vals[0], dU0, U[Opt.mode].nRows * U[Opt.mode].nCols * sizeof(DTYPE), hipMemcpyDeviceToHost), 0);

	hipFree(dVals); 
	hipFree(dU0); hipFree(dU1); hipFree(dU2); hipFree(dU3);
	hipFree(dfbrIdx0); hipFree(dInds2); hipFree(dInds3); 
	hipFree(dfbrIdx0); hipFree(dfbrIdx1); hipFree(dFbrIdx2);
	hipFree(dfbrPtr0); hipFree(dfbrPtr1); hipFree(dFbrPtr2);
	hipFree(dFbrLikeSlcInds);

	return 0;
}